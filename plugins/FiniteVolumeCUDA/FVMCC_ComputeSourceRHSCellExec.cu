#include "hip/hip_runtime.h"
#include "FiniteVolumeCUDA/FVMCC_ComputeSourceRHSCellExec.hh"
#include "Framework/MeshData.hh"
#include "Framework/CellConn.hh"
#include "Config/ConfigOptionPtr.hh"
#include "Framework/CudaDeviceManager.hh"
#include "Common/SelfRegistPtr.hh"
#include "Common/CUDA/CFVec.hh"
#include "Framework/CudaTimer.hh"
#include "FiniteVolume/FluxData.hh"
#include "FiniteVolume/KernelData.hh"
#include "FiniteVolume/CellData.hh"

#include "FiniteVolumeCUDA/FiniteVolumeCUDA.hh"
#include "Framework/MethodCommandProvider.hh"
#include "Framework/VarSetListT.hh"
#include "FiniteVolume/LaxFriedFlux.hh"
#include "FiniteVolume/LeastSquareP1PolyRec2D.hh"
#include "FiniteVolume/LeastSquareP1PolyRec3D.hh"
#include "FiniteVolume/BarthJesp.hh"
#include "FiniteVolume/Venktn2D.hh"
#include "MHD/MHD2DProjectionConsT.hh"
#include "MHD/MHD3DProjectionConsT.hh"
#include "MHD/MHD2DProjectionPrimT.hh"
#include "MHD/MHD3DProjectionPrimT.hh"
#include "MHD/MHDProjectionPrimToConsT.hh"
#include "FiniteVolumeMHD/LaxFriedFluxTanaka.hh"
#include "MHD/MHD2DProjectionVarSet.hh"   
#include "MHD/MHD3DProjectionVarSet.hh"

#include "Maxwell/Maxwell2DProjectionVarSet.hh"
#include "Maxwell/Maxwell2DProjectionConsT.hh"
#include "FiniteVolumeMaxwell/StegerWarmingMaxwellProjection2D.hh"

#include "MultiFluidMHD/MultiFluidMHDVarSet.hh"
#include "MultiFluidMHD/EulerMFMHD2DHalfConsT.hh"
#include "MultiFluidMHD/EulerMFMHD2DHalfRhoiViTiT.hh"
#include "MultiFluidMHD/EulerMFMHD2DHalfRhoiViTiToConsT.hh"
#include "MultiFluidMHD/EulerMFMHD2DHalfConsToRhoiViTiT.hh"
#include "MultiFluidMHD/EulerMFMHD2DConsT.hh"
#include "MultiFluidMHD/EulerMFMHD2DRhoiViTiT.hh"
#include "MultiFluidMHD/EulerMFMHD2DRhoiViTiToConsT.hh"
#include "MultiFluidMHD/EulerMFMHD2DConsToRhoiViTiT.hh"
#include "FiniteVolumeMultiFluidMHD/AUSMPlusUpFluxMultiFluid.hh"
#include "FiniteVolumeMultiFluidMHD/AUSMFluxMultiFluid.hh"
#include "FiniteVolumeMultiFluidMHD/DriftWaves2DHalfTwoFluid.hh"
#include "FiniteVolumeMultiFluidMHD/HartmannSourceTerm.hh"

//////////////////////////////////////////////////////////////////////////////

using namespace COOLFluiD::Framework;
using namespace COOLFluiD::Common;
using namespace COOLFluiD::Config;
using namespace COOLFluiD::Physics::MHD;
using namespace COOLFluiD::Physics::Maxwell;
using namespace COOLFluiD::Physics::MultiFluidMHD;

//////////////////////////////////////////////////////////////////////////////

namespace COOLFluiD {

  namespace Numerics {

    namespace FiniteVolume {

//////////////////////////////////////////////////////////////////////////////


//Provider for AUSMPlusUpFlux with Source
#define FVMCC_MULTIFLUIDMHD_RHS_PROV_AUSMPLUSUP_SOURCE(__dim__,__half__,__svars__,__uvars__,__sourceterm__,__limiter__,__nbBThreads__,__providerName__) \
MethodCommandProvider<FVMCC_ComputeSourceRHSCellExec<AUSMPlusUpFluxMultiFluid<MultiFluidMHDVarSet<Maxwell##__dim__##ProjectionVarSet> >, \
			              VarSetListT<EulerMFMHD##__dim__##__half__##__svars__##T, EulerMFMHD##__dim__##__half__##__uvars__##T>, \
				      __sourceterm__<MultiFluidMHDVarSet<Maxwell##__dim__##ProjectionVarSet> >, \
				      LeastSquareP1PolyRec##__dim__ , __limiter__, __nbBThreads__>, \
		      CellCenterFVMData, FiniteVolumeCUDAModule>	\
fvmcc_RhsMultiFluidMHD##__limiter__##AUSMPlusUp##__dim__##__half__##__svars__##__uvars__##__sourceterm__##__nbBThreads__##Provider(__providerName__);

// 48 block threads (default)
FVMCC_MULTIFLUIDMHD_RHS_PROV_AUSMPLUSUP_SOURCE(2D,Half,Cons,RhoiViTi,DriftWaves2DHalfTwoFluid,BarthJesp,48,"CellBarthAUSMPlusUpEulerMFMHD2DHalfRhoiViTiDriftWavesTwoFluid")
FVMCC_MULTIFLUIDMHD_RHS_PROV_AUSMPLUSUP_SOURCE(2D,Half,Cons,RhoiViTi,DriftWaves2DHalfTwoFluid,Venktn2D,48,"CellVenktnAUSMPlusUpEulerMFMHD2DHalfRhoiViTiHartmann")
#undef FVMCC_MULTIFLUIDMHD_RHS_PROV_AUSMPLUSUP_SOURCE

//////////////////////////////////////////////////////////////////////////////

template <typename PHYS>
HOST_DEVICE inline void setState(CFreal* state, CFreal* statePtr, 
				 CFreal* node, CFreal* nodePtr)
{
  // copy the state node data to shared memory
  for (CFuint i = 0; i < PHYS::DIM; ++i) {node[i] = nodePtr[i];}
  // copy the state data to shared memory
  for (CFuint i = 0; i < PHYS::NBEQS; ++i) {state[i] = statePtr[i];} 
}
      
//////////////////////////////////////////////////////////////////////////////
      
template <typename PHYS>
HOST_DEVICE inline void setFaceNormal(FluxData<PHYS>* fd, CFreal* normal)
{
  CudaEnv::CFVecSlice<CFreal,PHYS::DIM> n(normal);
  const CFreal area = n.norm2();
  fd->setFaceArea(area);
  const CFreal ovArea = 1./area;
  CudaEnv::CFVecSlice<CFreal,PHYS::DIM> un(fd->getUnitNormal());
  for (CFuint i = 0; i < PHYS::DIM; ++i) {
    un[i] = n[i]*ovArea;
  }
}
      
//////////////////////////////////////////////////////////////////////////////

template <typename PHYS, typename PTR>
HOST_DEVICE void setFluxData(const CFuint f, const CFint stype, 
			     const CFuint stateID, const CFuint cellID, 
			     KernelData<CFreal>* kd, FluxData<PHYS>* fd,
			     PTR cellFaces)
{
  fd->setStateID(RIGHT, stateID);
  CFreal* statePtrR = (stype > 0) ? &kd->states[stateID*PHYS::NBEQS] : &kd->ghostStates[stateID*PHYS::NBEQS];  
  CFreal* nodePtrR = (stype > 0) ? &kd->centerNodes[stateID*PHYS::DIM] : &kd->ghostNodes[stateID*PHYS::DIM];  
  setState<PHYS>(fd->getState(RIGHT), statePtrR, fd->getNode(RIGHT), nodePtrR);
  
  fd->setIsBFace(stype < 0);
  fd->setStateID(LEFT, cellID);
  const CFuint faceID = cellFaces[f*kd->nbCells + cellID];
  fd->setIsOutward(kd->isOutward[faceID] == cellID);
  
  CFreal* statePtrL = &kd->states[cellID*PHYS::NBEQS];
  CFreal* nodePtrL = &kd->centerNodes[cellID*PHYS::DIM];
  setState<PHYS>(fd->getState(LEFT), statePtrL, fd->getNode(LEFT), nodePtrL);
  setFaceNormal<PHYS>(fd, &kd->normals[faceID*PHYS::DIM]);
}

//////////////////////////////////////////////////////////////////////////////

template <typename T, CFuint SIZE>
void print(const std::string& name, T* array) 
{
  CFLog(INFO, name << " = \t");
  for (CFuint i = 0; i < SIZE; ++i) {
    CFLog(INFO, array[i] << " ");
  }
  CFLog(INFO, "\n");
}

//////////////////////////////////////////////////////////////////////////////

template <typename MODEL>
HOST_DEVICE void computeFaceCentroid(const CellData::Itr* cell, const CFuint faceIdx, 
				     const CFreal* nodes, CFreal* midFaceCoord)
{  
  CudaEnv::CFVecSlice<CFreal, MODEL::DIM> coord(midFaceCoord);
  coord = 0.;
  const CFuint nbFaceNodes = cell->getNbFaceNodes(faceIdx);
  const CFreal ovNbFaceNodes = 1./(static_cast<CFreal>(nbFaceNodes));
  for (CFuint n = 0; n < nbFaceNodes; ++n) {
    const CFuint cellNodeID = cell->getNodeID(faceIdx, n);
    const CFuint nodeID = cell->getNodeID(faceIdx,n);
    const CFreal* faceNode = &nodes[nodeID*MODEL::DIM];
    for (CFuint d = 0; d < MODEL::DIM; ++d) {
      coord[d] += faceNode[d];
    }
  }
  coord *= ovNbFaceNodes;
}

//////////////////////////////////////////////////////////////////////////////

template <typename PHYS, typename POLYREC>
__global__ void computeGradientsKernel(typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
				       const CFuint nbCells,
				       CFreal* states, 
				       CFreal* nodes,
				       CFreal* centerNodes,
				       CFreal* ghostStates,
				       CFreal* ghostNodes,
				       CFreal* uX,
				       CFreal* uY,
				       CFreal* uZ,
				       CFreal* limiter,
				       CFreal* updateCoeff, 
				       CFreal* rhs,
				       CFreal* normals,
				       CFint* isOutward,
				       const CFuint* cellInfo,
				       const CFuint* cellStencil,
				       const CFuint* cellFaces,
				       const CFuint* cellNodes,
				       const CFint*  neighborTypes,
				       const Framework::CellConn* cellConn)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
  // dcor->init(gdcor);
  
  if (cellID < nbCells) {    
    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
			   rhs, normals, uX, uY, uZ, isOutward);
    
    // compute and store cell gradients at once 
    POLYREC polyRec(dcor);
    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
    CellData::Itr cell = cells.getItr(cellID);
    polyRec.computeGradients(&states[cellID*PHYS::NBEQS], &centerNodes[cellID*PHYS::DIM], &kd, &cell);
  }
}
      
//////////////////////////////////////////////////////////////////////////////

template <typename PHYS, typename POLYREC, typename LIMITER>
__global__ void computeLimiterKernel(typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol,
				     typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
				     const CFuint nbCells,
				     CFreal* states, 
				     CFreal* nodes,
				     CFreal* centerNodes,
				     CFreal* ghostStates,
				     CFreal* ghostNodes,
				     CFreal* uX,
				     CFreal* uY,
				     CFreal* uZ,
				     CFreal* limiter,
				     CFreal* updateCoeff, 
				     CFreal* rhs,
				     CFreal* normals,
				     CFint* isOutward,
				     const CFuint* cellInfo,
				     const CFuint* cellStencil,
				     const CFuint* cellFaces,
				     const CFuint* cellNodes,
				     const CFint*  neighborTypes,
				     const Framework::CellConn* cellConn)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  // __shared__ typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE> s_dcol[32];
  // typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol = &s_dcol[threadIdx.x];
  // dcol->init(gdcol);
  
  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
  // dcor->init(gdcor);
  
  if (cellID < nbCells) {    
    // compute all cell quadrature points at once (size of this array is overestimated)
    CFreal midFaceCoord[PHYS::DIM*PHYS::DIM*2];
    
    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
    CellData::Itr cell = cells.getItr(cellID);
    const CFuint nbFacesInCell = cell.getNbFacesInCell();
    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
      computeFaceCentroid<PHYS>(&cell, f, nodes, &midFaceCoord[f*PHYS::DIM]);
    }
    
    // compute cell-based limiter at once
    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
			   rhs, normals, uX, uY, uZ, isOutward);
    LIMITER limt(dcol);
    
    if (dcor->currRes > dcor->limitRes && (dcor->limitIter > 0 && dcor->currIter < dcor->limitIter)) {	
      limt.limit(&kd, &cell, &midFaceCoord[0], &limiter[cellID*PHYS::NBEQS]);
    }
    else {
      if (!dcor->freezeLimiter) {
	// historical modification of the limiter
	CudaEnv::CFVec<CFreal,PHYS::NBEQS> tmpLimiter;
	limt.limit(&kd, &cell, &midFaceCoord[0], &tmpLimiter[0]);
	CFuint currID = cellID*PHYS::NBEQS;
	for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar, ++currID) {
	  limiter[currID] = min(tmpLimiter[iVar],limiter[currID]);
	}
      }
    }
  }
}
  
//////////////////////////////////////////////////////////////////////////////
             
template <typename SCHEME, typename POLYREC>
__global__ void computeFluxKernel(typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof,
				  typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
				  typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
				  const CFuint nbCells,
				  CFreal* states, 
				  CFreal* nodes,
				  CFreal* centerNodes,
				  CFreal* ghostStates,
				  CFreal* ghostNodes,
				  CFreal* uX,
				  CFreal* uY,
				  CFreal* uZ,
				  CFreal* limiter,
				  CFreal* updateCoeff, 
				  CFreal* rhs,
				  CFreal* normals,
				  CFint* isOutward,
				  const CFuint* cellInfo,
				  const CFuint* cellStencil,
				  const CFuint* cellFaces,
				  const CFuint* cellNodes,
				  const CFint*  neighborTypes,
				  const Framework::CellConn* cellConn)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  // __shared__ typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE> s_dcof[32];
  // typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof = &s_dcof[threadIdx.x];
  // dcof->init(gdcof);
  
  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
  // dcor->init(gdcor);
  
  // __shared__ typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE> s_dcop[32];
  // typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop = &s_dcop[threadIdx.x];
  // dcop->init(gdcop);
  
  if (cellID < nbCells) {
    // reset the rhs and update coefficients to 0
    CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> res(&rhs[cellID*SCHEME::MODEL::NBEQS]);
    res = 0.;
    updateCoeff[cellID] = 0.;
    
    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
			   rhs, normals, uX, uY, uZ, isOutward);
    
    // compute and store cell gradients at once 
    POLYREC polyRec(dcor);
    SCHEME fluxScheme(dcof);
    CFreal midFaceCoord[SCHEME::MODEL::DIM*SCHEME::MODEL::DIM*2];
    FluxData<typename SCHEME::MODEL> currFd; currFd.initialize();
    typename SCHEME::MODEL pmodel(dcop);
    
    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
    CellData::Itr cell = cells.getItr(cellID);
    
    // compute the fluxes
    const CFuint nbFacesInCell = cell.getNbActiveFacesInCell();
    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
      const CFint stype = cell.getNeighborType(f);
      
      if (stype != 0) { // skip all partition faces
	// set all flux data for the current face
	const CFuint stateID = cell.getNeighborID(f);
	setFluxData(f, stype, stateID, cellID, &kd, &currFd, cellFaces);
	
	// compute face quadrature points (centroid)
	CFreal* faceCenters = &midFaceCoord[f*SCHEME::MODEL::DIM];
	computeFaceCentroid<typename SCHEME::MODEL>(&cell, f, nodes, faceCenters);
	
	// extrapolate solution on quadrature points on both sides of the face
	polyRec.extrapolateOnFace(&currFd, faceCenters, uX, uY, uZ, limiter);
	
	// compute the convective flux across the face
        fluxScheme.prepareComputation(&currFd, &pmodel);
	fluxScheme(&currFd, &pmodel);
	
	// update the residual
	CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> ress(currFd.getResidual());
	res -= ress;
	
	// update the update coefficient
	updateCoeff[cellID] += currFd.getUpdateCoeff();
      }
    }
  }
}
 


//////////////////////////////////////////////////////////////////////////////

template <typename SOURCE>
__global__ void computeSource(typename SOURCE::BASE::template DeviceConfigOptions<NOTYPE>* dcos,
				  typename SOURCE::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
				  const CFuint nbCells,
				  CFreal* states, 
                                  CFreal* volumes,
				  CFreal* nodes,
				  CFreal* centerNodes,
				  CFreal* ghostStates,
				  CFreal* ghostNodes,
				  CFreal* uX,
				  CFreal* uY,
				  CFreal* uZ,
				  CFreal* limiter,
				  CFreal* updateCoeff, 
				  CFreal* rhs,
				  CFreal* normals,
				  CFint* isOutward,
				  const CFuint* cellInfo,
				  const CFuint* cellStencil,
				  const CFuint* cellFaces,
				  const CFuint* cellNodes,
				  const CFint*  neighborTypes,
				  const Framework::CellConn* cellConn,
				  CFreal ResFactor, bool IsAxisymmetric)
{
  // each thread takes care of computing the source for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;

  const CFuint nbEqs = SOURCE::MODEL::NBEQS;
  CudaEnv::CFVec<CFreal,SOURCE::MODEL::NBEQS> source;
  source = 0.;

  SOURCE Source(dcos);
  typename SOURCE::MODEL pmodel(dcop);

  CudaEnv::CFVecSlice<CFreal,SOURCE::MODEL::NBEQS> state(&states[cellID*SOURCE::MODEL::NBEQS]);
  Source(&state[0], &pmodel, &source[0]);
      
  CFreal invR = 1.0;
  if (IsAxisymmetric) {     
    //invR /= abs(currCell->getState(0)->getCoordinates()[YY]);  //Not implemented
  }


  for (CFuint iEq = 0; iEq < nbEqs; ++iEq) { 
     rhs[iEq] += ResFactor*source[iEq]*invR;   
  }
}



 
//////////////////////////////////////////////////////////////////////////////

template <typename SCHEME, typename SOURCE, typename POLYREC, typename LIMITER>
void computeFluxSourceCPU(typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof,
		    typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
		    typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol,
		    typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
                    typename SOURCE::BASE::template DeviceConfigOptions<NOTYPE>* dcos,
		    const CFuint nbCells,
		    CFreal* states, 
                    CFreal* volumes,
		    CFreal* nodes,
		    CFreal* centerNodes,
		    CFreal* ghostStates,
		    CFreal* ghostNodes,
		    CFreal* uX,
		    CFreal* uY,
		    CFreal* uZ,
		    CFreal* limiter,
		    CFreal* updateCoeff, 
		    CFreal* rhs,
		    CFreal* normals,
		    CFint* isOutward,
		    const CFuint* cellInfo,
		    const CFuint* cellStencil,
		    const CFuint* cellFaces,
		    const CFuint* cellNodes,
		    const CFint* neighborTypes,
		    const Framework::CellConn* cellConn,
                    CFreal ResFactor, bool IsAxisymmetric)
{ 
  typedef typename SCHEME::MODEL PHYS;
  
  FluxData<PHYS> fd; fd.initialize();
  FluxData<PHYS>* currFd = &fd;
  cf_assert(currFd != CFNULL);
  SCHEME fluxScheme(dcof);
  POLYREC polyRec(dcor);
  LIMITER limt(dcol);
  PHYS pmodel(dcop);
  
  CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
  KernelData<CFreal> kd(nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
			rhs, normals, uX, uY, uZ, isOutward);
  
  CFreal midFaceCoord[PHYS::DIM*PHYS::DIM*2];
  CudaEnv::CFVec<CFreal,PHYS::NBEQS> tmpLimiter;
  
  CudaEnv::CFVec<CFreal,PHYS::NBEQS> source;
  SOURCE Source(dcos);

  // compute the cell-based gradients
  for (CFuint cellID = 0; cellID < nbCells; ++cellID) {
    CellData::Itr cell = cells.getItr(cellID);
    polyRec.computeGradients(&states[cellID*PHYS::NBEQS], &centerNodes[cellID*PHYS::DIM], &kd, &cell);
  }
  
  // compute the cell based limiter
  // for (CFuint cellID = 0; cellID < nbCells; ++cellID) {
  for (CellData::Itr cell = cells.begin(); cell <= cells.end(); ++cell) {
    // compute all cell quadrature points at once (size of this array is overestimated)
    const CFuint nbFacesInCell = cell.getNbFacesInCell();
    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
      computeFaceCentroid<PHYS>(&cell, f, nodes, &midFaceCoord[f*PHYS::DIM]);
    }
    
    const CFuint cellID = cell.getCellID();
    if (dcor->currRes > dcor->limitRes && (dcor->limitIter > 0 && dcor->currIter < dcor->limitIter)) {	
      // compute cell-based limiter
      limt.limit(&kd, &cell, &midFaceCoord[0], &limiter[cellID*PHYS::NBEQS]);
    }
    else {
      if (!dcor->freezeLimiter) {
	// historical modification of the limiter
	limt.limit(&kd, &cell, &midFaceCoord[0], &tmpLimiter[0]);
	CFuint currID = cellID*PHYS::NBEQS;
	for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar, ++currID) {
	  limiter[currID] = min(tmpLimiter[iVar],limiter[currID]);
	}
      }
    }
  }
  
  // compute the fluxes
  for (CellData::Itr cell = cells.begin(); cell <= cells.end(); ++cell) {
    // reset the rhs and update coefficients to 0
    const CFuint cellID = cell.getCellID();
    CudaEnv::CFVecSlice<CFreal,PHYS::NBEQS> res(&rhs[cellID*PHYS::NBEQS]);
    res = 0.;
    updateCoeff[cellID] = 0.;
    
    const CFuint nbFacesInCell = cell.getNbActiveFacesInCell();
    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
      const CFint stype = cell.getNeighborType(f);
      
      if (stype != 0) { // skip all partition faces
	const CFuint stateID =  cell.getNeighborID(f);
	setFluxData(f, stype, stateID, cellID, &kd, currFd, cellFaces);
	
	// compute face quadrature points (centroid)
	CFreal* faceCenters = &midFaceCoord[f*PHYS::DIM];
	computeFaceCentroid<PHYS>(&cell, f, nodes, faceCenters);
	
	// extrapolate solution on quadrature points on both sides of the face
	polyRec.extrapolateOnFace(currFd, faceCenters, uX, uY, uZ, limiter);
        fluxScheme.prepareComputation(currFd, &pmodel);
	fluxScheme(currFd, &pmodel); // compute the convective flux across the face
	
	for (CFuint iEq = 0; iEq < PHYS::NBEQS; ++iEq) {
	  const CFreal value = currFd->getResidual()[iEq];
	  res[iEq] -= value;  // update the residual 
	}
	
	// update the update coefficient
	updateCoeff[cellID] += currFd->getUpdateCoeff();
      }
    }

 
    //Source computation
    source = 0.;
    
 
    CudaEnv::CFVecSlice<CFreal,PHYS::NBEQS> state(&states[cellID*PHYS::NBEQS]);
    Source(&state[0], &pmodel, &source[0]);

    CFreal invR = 1.0;
    if (IsAxisymmetric) {     
      //invR /= abs(currCell->getState(0)->getCoordinates()[YY]);  
    }
    CFreal factor = invR*volumes[cellID]*ResFactor;     

    source *= factor;
    for (CFuint iEq = 0; iEq < PHYS::NBEQS; ++iEq) { 
      res[iEq] += source[iEq];   
    }
    
    /////////////////////////////////////////////


  }
}

//////////////////////////////////////////////////////////////////////////////

template <typename SCHEME, typename PHYSICS, typename SOURCE,typename POLYREC, typename LIMITER, CFuint NB_BLOCK_THREADS>
void FVMCC_ComputeSourceRHSCellExec<SCHEME,PHYSICS,SOURCE,POLYREC,LIMITER,NB_BLOCK_THREADS>::execute()
{
  using namespace COOLFluiD::Framework;
  using namespace COOLFluiD::Common;
  
  CFTRACEBEGIN;
  
  CFLog(VERBOSE, "FVMCC_ComputeSourceRHSCellExec::execute() START\n");
  
  this->initializeComputationRHS();
  
  const CFuint nbCells = this->socket_states.getDataHandle().size();
  cf_assert(nbCells > 0);
  DataHandle<CFreal> updateCoeff = this->socket_updateCoeff.getDataHandle();
  DataHandle<CFreal> rhs = this->socket_rhs.getDataHandle();
  DataHandle<CFreal> normals = this->socket_normals.getDataHandle();
  DataHandle<CFint> isOutward = this->socket_isOutward.getDataHandle();  
  
  SafePtr<SCHEME> lf  = this->getMethodData().getFluxSplitter().template d_castTo<SCHEME>();
  SafePtr<POLYREC> pr = this->getMethodData().getPolyReconstructor().template d_castTo<POLYREC>();
  SafePtr<LIMITER> lm = this->getMethodData().getLimiter().template d_castTo<LIMITER>();
  SafePtr<typename PHYSICS::PTERM> phys = PhysicalModelStack::getActive()->getImplementor()->
    getConvectiveTerm().template d_castTo<typename PHYSICS::PTERM>();
  
  typedef typename SCHEME::template DeviceFunc<GPU, PHYSICS> FluxScheme;  
  typedef typename POLYREC::template DeviceFunc<PHYSICS> PolyRec;  
  typedef typename LIMITER::template DeviceFunc<PHYSICS> Limiter;  
  
  //Added for Source
  SelfRegistPtr<SOURCE> ls1  = (*this->getMethodData().getSourceTermComputer())[0].template d_castTo<SOURCE>();  //Only valid if there is only one source term!!
  SafePtr<SOURCE> ls = ls1.getPtr();
  typedef typename SOURCE::template DeviceFunc<GPU, PHYSICS> SourceTerm; 
  
  if (this->m_onGPU) {
    
    CudaEnv::CudaTimer& timer = CudaEnv::CudaTimer::getInstance();
    timer.start();
    
    // copy of data that change at every iteration
    this->socket_states.getDataHandle().getGlobalArray()->put();
    this->socket_volumes.getDataHandle().getLocalArray()->put(); 
    this->m_ghostStates.put();
    
    CFLog(VERBOSE, "FVMCC_ComputeSourceRHSCellExec::execute() => CPU-->GPU data transfer took " << timer.elapsed() << " s\n");
    timer.start();
    
    ConfigOptionPtr<POLYREC, NOTYPE, GPU> dcor(pr);
    ConfigOptionPtr<LIMITER, NOTYPE, GPU> dcol(lm);
    ConfigOptionPtr<SCHEME,  NOTYPE, GPU> dcof(lf);
    ConfigOptionPtr<typename PHYSICS::PTERM, NOTYPE, GPU> dcop(phys);
    
    //Added for Source    
    ConfigOptionPtr<SOURCE, NOTYPE, GPU> dcos(ls);
    
    const CFuint blocksPerGrid = CudaEnv::CudaDeviceManager::getInstance().getBlocksPerGrid(nbCells);
    const CFuint nThreads = CudaEnv::CudaDeviceManager::getInstance().getNThreads();
    
    //dim3 blocks(m_nbBlocksPerGridX, m_nbBlocksPerGridY);
    
    //hipFuncSetCacheConfig(reinterpret_cast<const void*>("computeGradientsKernel"), hipFuncCachePreferL1);
    
    // compute the cell-based gradients
    computeGradientsKernel<PHYSICS, PolyRec> <<<blocksPerGrid,nThreads>>> 
      (dcor.getPtr(),
       nbCells,
       this->socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
       this->socket_nodes.getDataHandle().getGlobalArray()->ptrDev(),
       this->m_centerNodes.ptrDev(), 
       this->m_ghostStates.ptrDev(),
       this->m_ghostNodes.ptrDev(),
       this->socket_uX.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_uY.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_uZ.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_limiter.getDataHandle().getLocalArray()->ptrDev(),
       updateCoeff.getLocalArray()->ptrDev(), 
       rhs.getLocalArray()->ptrDev(),
       normals.getLocalArray()->ptrDev(),
       isOutward.getLocalArray()->ptrDev(),
       this->m_cellInfo.ptrDev(),
       this->m_cellStencil.ptrDev(),
       this->m_cellFaces->getPtr()->ptrDev(),
       this->m_cellNodes->getPtr()->ptrDev(),
       this->m_neighborTypes.ptrDev(),
       this->m_cellConn.ptrDev());
    
    CFLog(VERBOSE, "FVMCC_ComputeSourceRHSCellExec::execute() => computeGradientsKernel took " << timer.elapsed() << " s\n");
    
    timer.start();
    
    // hipFuncSetCacheConfig(reinterpret_cast<const void*>("computeLimiterKernel"), hipFuncCachePreferL1);
    
    // compute the limiter in each cell
    computeLimiterKernel<PHYSICS, PolyRec, Limiter> <<<blocksPerGrid,nThreads>>> 
      (dcol.getPtr(),
       dcor.getPtr(),
       nbCells,
       this->socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
       this->socket_nodes.getDataHandle().getGlobalArray()->ptrDev(),
       this->m_centerNodes.ptrDev(), 
       this->m_ghostStates.ptrDev(),
       this->m_ghostNodes.ptrDev(),
       this->socket_uX.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_uY.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_uZ.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_limiter.getDataHandle().getLocalArray()->ptrDev(),
       updateCoeff.getLocalArray()->ptrDev(), 
       rhs.getLocalArray()->ptrDev(),
       normals.getLocalArray()->ptrDev(),
       isOutward.getLocalArray()->ptrDev(),
       this->m_cellInfo.ptrDev(),
       this->m_cellStencil.ptrDev(),
       this->m_cellFaces->getPtr()->ptrDev(),
       this->m_cellNodes->getPtr()->ptrDev(),
       this->m_neighborTypes.ptrDev(),
       this->m_cellConn.ptrDev());
    
    CFLog(VERBOSE, "FVMCC_ComputeSourceRHSCellExec::execute() => computeLimiterKernel took " << timer.elapsed() << " s\n");
    
    timer.start();
    
    // hipFuncSetCacheConfig(reinterpret_cast<const void*>("computeFluxKernel"), hipFuncCachePreferL1);
    
    // compute the convective flux in each cell
    computeFluxKernel<FluxScheme, PolyRec> <<<blocksPerGrid,nThreads>>> 
      (dcof.getPtr(),
       dcor.getPtr(),
       dcop.getPtr(),
       nbCells,
       this->socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
       this->socket_nodes.getDataHandle().getGlobalArray()->ptrDev(),
       this->m_centerNodes.ptrDev(), 
       this->m_ghostStates.ptrDev(),
       this->m_ghostNodes.ptrDev(),
       this->socket_uX.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_uY.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_uZ.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_limiter.getDataHandle().getLocalArray()->ptrDev(),
       updateCoeff.getLocalArray()->ptrDev(), 
       rhs.getLocalArray()->ptrDev(),
       normals.getLocalArray()->ptrDev(),
       isOutward.getLocalArray()->ptrDev(),
       this->m_cellInfo.ptrDev(),
       this->m_cellStencil.ptrDev(),
       this->m_cellFaces->getPtr()->ptrDev(),
       this->m_cellNodes->getPtr()->ptrDev(),
       this->m_neighborTypes.ptrDev(),
       this->m_cellConn.ptrDev());
    
    CFLog(VERBOSE, "FVMCC_ComputeSourceRHSCellExec::execute() => computeFluxKernel took " << timer.elapsed() << " s\n");

    timer.start();
    CFLog(VERBOSE, "FVMCC_ComputeRHS::execute() => before computeSourceTerm()\n");

    bool IsAxisymmetric = this->getMethodData().isAxisymmetric(); //Default = false
    CFreal ResFactor = this->getMethodData().getResFactor(); //Default = 1

    computeSource<SourceTerm> <<<blocksPerGrid,nThreads>>> 
      (dcos.getPtr(),
       dcop.getPtr(),
       nbCells,
       this->socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
       this->socket_volumes.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_nodes.getDataHandle().getGlobalArray()->ptrDev(),
       this->m_centerNodes.ptrDev(), 
       this->m_ghostStates.ptrDev(),
       this->m_ghostNodes.ptrDev(),
       this->socket_uX.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_uY.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_uZ.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_limiter.getDataHandle().getLocalArray()->ptrDev(),
       updateCoeff.getLocalArray()->ptrDev(), 
       rhs.getLocalArray()->ptrDev(),
       normals.getLocalArray()->ptrDev(),
       isOutward.getLocalArray()->ptrDev(),
       this->m_cellInfo.ptrDev(),
       this->m_cellStencil.ptrDev(),
       this->m_cellFaces->getPtr()->ptrDev(),
       this->m_cellNodes->getPtr()->ptrDev(),
       this->m_neighborTypes.ptrDev(),
       this->m_cellConn.ptrDev(),
       ResFactor, IsAxisymmetric);

    CFLog(VERBOSE, "FVMCC_ComputeRHS::execute() => computeSourceTerm took " << timer.elapsed() << "\n");
    
    timer.start();
    rhs.getLocalArray()->get();
    updateCoeff.getLocalArray()->get();
    CFLog(VERBOSE, "FVMCC_ComputeSourceRHSCellExec::execute() => GPU-->CPU data transfer took " << timer.elapsed() << " s\n");
  }
  else {
    // AL: useful fo debugging
    // for (CFuint i = 0; i <  m_ghostStates.size()/9; ++i) {
    //   std::cout.precision(12); std::cout << "g" << i << " => ";
    //   for (CFuint j = 0; j < 9; ++j) {
    // 	std::cout << m_ghostStates[i*9+j] << " ";
    //   }
    //   std::cout << "\n";
    // }
    // for (CFuint i = 0; i <  socket_states.getDataHandle().size(); ++i) {
    //   std::cout.precision(12); std::cout << i << " => "<< *socket_states.getDataHandle()[i] <<"\n";
    // }
    
    ConfigOptionPtr<SCHEME>  dcof(lf);
    ConfigOptionPtr<POLYREC> dcor(pr);
    ConfigOptionPtr<LIMITER> dcol(lm);
    ConfigOptionPtr<typename PHYSICS::PTERM> dcop(phys);
    ConfigOptionPtr<SOURCE> dcos(ls);

    bool IsAxisymmetric = this->getMethodData().isAxisymmetric(); //Default = false
    CFreal ResFactor = this->getMethodData().getResFactor(); //Default = 1

    computeFluxSourceCPU<FluxScheme, SourceTerm, PolyRec, Limiter>
      (dcof.getPtr(),
       dcor.getPtr(),
       dcol.getPtr(),
       dcop.getPtr(),
       dcos.getPtr(),
       nbCells,
       this->socket_states.getDataHandle().getGlobalArray()->ptr(), 
       this->socket_volumes.getDataHandle().getLocalArray()->ptr(),
       this->socket_nodes.getDataHandle().getGlobalArray()->ptr(),
       this->m_centerNodes.ptr(), 
       this->m_ghostStates.ptr(),
       this->m_ghostNodes.ptr(),
       this->socket_uX.getDataHandle().getLocalArray()->ptr(),
       this->socket_uY.getDataHandle().getLocalArray()->ptr(),
       this->socket_uZ.getDataHandle().getLocalArray()->ptr(),
       this->socket_limiter.getDataHandle().getLocalArray()->ptr(),
       updateCoeff.getLocalArray()->ptr(), 
       rhs.getLocalArray()->ptr(),
       normals.getLocalArray()->ptr(),
       isOutward.getLocalArray()->ptr(),
       this->m_cellInfo.ptr(),
       this->m_cellStencil.ptr(),
       this->m_cellFaces->getPtr()->ptr(),
       this->m_cellNodes->getPtr()->ptr(),
       this->m_neighborTypes.ptr(),
       this->m_cellConn.ptr(),
       ResFactor, IsAxisymmetric);
  }
  
// for (int i = 0; i < updateCoeff.size(); ++i) {
//      std::cout << "updateCoeff[" << i << "] = " << updateCoeff[i]  << std::endl;
//       /* std::cout << "rhs[" << i << "] = ";
//        for (int j = 0; j < 9; ++j) {
//          std::cout << rhs[i*9+j] << " ";
//        }
//        std::cout << std::endl;*/
// } 
//   abort();
  
  this->finalizeComputationRHS();
  
  CFLog(VERBOSE, "FVMCC_ComputeSourceRHSCellExec::execute() END\n");
  
  CFTRACEEND;
}

//////////////////////////////////////////////////////////////////////////////

    } // namespace FiniteVolume
    
  } // namespace Numerics

} // namespace COOLFluiD
