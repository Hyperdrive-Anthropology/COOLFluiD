#include "hip/hip_runtime.h"
#include "FluxReconstructionCUDA/ConvDiffRHSFluxReconstructionCUDA.hh"
#include "Framework/MeshData.hh"
#include "Framework/CellConn.hh"
#include "Config/ConfigOptionPtr.hh"
#include "Framework/CudaDeviceManager.hh"
#include "Common/CUDA/CFVec.hh"
#include "Framework/CudaTimer.hh"

#include "FluxReconstructionMethod/FluxData.hh"
#include "FluxReconstructionMethod/KernelData.hh"
#include "FluxReconstructionMethod/CellData.hh"

#include "FluxReconstructionCUDA/FluxReconstructionCUDA.hh"
#include "Framework/MethodCommandProvider.hh"
#include "Framework/VarSetListT.hh"
#include "Framework/MathTypes.hh"
#include "NavierStokes/Euler2DVarSetT.hh"
#include "NavierStokes/Euler2DConsT.hh"
#include "NavierStokes/NavierStokes2DVarSetT.hh"
#include "NavierStokes/NavierStokes2DConsT.hh"
#include "NavierStokes/NSVarSetListT.hh"

#include "FluxReconstructionMethod/LaxFriedrichsFlux.hh"
#include <stdio.h>

//////////////////////////////////////////////////////////////////////////////

using namespace COOLFluiD::Framework;
using namespace COOLFluiD::Common;
using namespace COOLFluiD::Config;
using namespace COOLFluiD::Physics::NavierStokes;

//////////////////////////////////////////////////////////////////////////////

namespace COOLFluiD {

    namespace FluxReconstructionMethod {

//////////////////////////////////////////////////////////////////////////////

#define FR_NS_RHS_PROV(__dim__,__svars__,__uvars__,__order__,__nbBThreads__,__providerName__) \
MethodCommandProvider<ConvDiffRHSFluxReconstructionCUDA<LaxFriedrichsFlux, \
                      VarSetListT<Euler##__dim__##__svars__##T, Euler##__dim__##__uvars__##T>,NSVarSetListT<NavierStokes##__dim__##__svars__##T, NavierStokes##__dim__##__uvars__##T>,__order__,__nbBThreads__>, \
		      FluxReconstructionSolverData,FluxReconstructionCUDAModule>	\
FR_RhsNS##__dim__##__svars__##__uvars__##__order__##__nbBThreads__##Provider(__providerName__);
// 48 block threads (default)
FR_NS_RHS_PROV(2D, Cons, Cons, 0, 48, "NSFRLaxFriedrichs2DConsP0")
FR_NS_RHS_PROV(2D, Cons, Cons, 1, 48, "NSFRLaxFriedrichs2DConsP1")
FR_NS_RHS_PROV(2D, Cons, Cons, 2, 48, "NSFRLaxFriedrichs2DConsP2")
FR_NS_RHS_PROV(2D, Cons, Cons, 3, 48, "NSFRLaxFriedrichs2DConsP3")
FR_NS_RHS_PROV(2D, Cons, Cons, 4, 48, "NSFRLaxFriedrichs2DConsP4")
FR_NS_RHS_PROV(2D, Cons, Cons, 5, 48, "NSFRLaxFriedrichs2DConsP5")
FR_NS_RHS_PROV(2D, Cons, Cons, 6, 48, "NSFRLaxFriedrichs2DConsP6")
FR_NS_RHS_PROV(2D, Cons, Cons, 7, 48, "NSFRLaxFriedrichs2DConsP7")
FR_NS_RHS_PROV(2D, Cons, Cons, 8, 48, "NSFRLaxFriedrichs2DConsP8")
FR_NS_RHS_PROV(2D, Cons, Cons, 9, 48, "NSFRLaxFriedrichs2DConsP9")
//FR_EULER_RHS_PROV(3D, Cons, Cons, 48, "EulerFRLaxFried3DCons")
//FR_NS_RHS_PROV(2D, ProjectionCons, ProjectionPrim, 48, "CellLaxFriedMHD2DPrim")
//FR_NS_RHS_PROV(3D, ProjectionCons, ProjectionPrim, 48, "CellLaxFriedMHD3DPrim")
#undef FR_NS_RHS_PROV

//////////////////////////////////////////////////////////////////////////////

template <typename PHYS>
HOST_DEVICE inline void setState(CFreal* state, CFreal* statePtr)
{
  // copy the state node data to shared memory
  //for (CFuint i = 0; i < PHYS::DIM; ++i) {node[i] = nodePtr[i];}
  // copy the state data to shared memory
  for (CFuint i = 0; i < PHYS::NBEQS; ++i) {state[i] = statePtr[i];} 
}
      
//////////////////////////////////////////////////////////////////////////////

template <typename PHYS, CFuint ORDER>
HOST_DEVICE void setFluxData(const CFuint stateID, const CFuint cellID, 
			     KernelData<CFreal>* kd, FluxData<PHYS,ORDER>* fd, const CFuint iSol)
{
  fd->setStateID(LEFT, stateID);
  CFreal* statePtrR = &kd->states[stateID*PHYS::NBEQS];  

  setState<PHYS>(fd->getState(iSol), statePtrR);

  fd->setNbSolPnts(kd->nbSolPnts);
}

//////////////////////////////////////////////////////////////////////////////

template <typename T, CFuint SIZE>
void print(const std::string& name, T* array) 
{
  CFLog(INFO, name << " = \t");
  for (CFuint i = 0; i < SIZE; ++i) {
    CFLog(INFO, array[i] << " ");
  }
  CFLog(INFO, "\n");
}

//////////////////////////////////////////////////////////////////////////////

//template <typename MODEL>
//HOST_DEVICE void computeFaceCentroid(const CellData::Itr* cell, const CFuint faceIdx, 
//				     const CFreal* nodes, CFreal* midFaceCoord)
//{  
//  CudaEnv::CFVecSlice<CFreal, MODEL::DIM> coord(midFaceCoord);
//  coord = 0.;
//  const CFuint nbFaceNodes = cell->getNbFaceNodes(faceIdx);
//  const CFreal ovNbFaceNodes = 1./(static_cast<CFreal>(nbFaceNodes));
//  for (CFuint n = 0; n < nbFaceNodes; ++n) {
//    const CFuint cellNodeID = cell->getNodeID(faceIdx, n);
//    const CFuint nodeID = cell->getNodeID(faceIdx,n);
//    const CFreal* faceNode = &nodes[nodeID*MODEL::DIM];
//    for (CFuint d = 0; d < MODEL::DIM; ++d) {
//      coord[d] += faceNode[d];
//    }
//  }
//  coord *= ovNbFaceNodes;
//}

//////////////////////////////////////////////////////////////////////////////

//template <typename PHYS, typename POLYREC>
//__global__ void computeGradientsKernel(typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
//				       const CFuint nbCells,
//				       CFreal* states, 
//				       CFreal* nodes,
//				       CFreal* centerNodes,
//				       CFreal* ghostStates,
//				       CFreal* ghostNodes,
//				       CFreal* uX,
//				       CFreal* uY,
//				       CFreal* uZ,
//				       CFreal* limiter,
//				       CFreal* updateCoeff, 
//				       CFreal* rhs,
//				       CFreal* normals,
//				       CFint* isOutward,
//				       const CFuint* cellInfo,
//				       const CFuint* cellStencil,
//				       const CFuint* cellFaces,
//				       const CFuint* cellNodes,
//				       const CFint*  neighborTypes,
//				       const Framework::CellConn* cellConn)
//{    
//  // each thread takes care of computing the gradient for one single cell
//  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
//  
//  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
//  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
//  // dcor->init(gdcor);
//  
//  if (cellID < nbCells) {    
//    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
//			   rhs, normals, uX, uY, uZ, isOutward);
//    
//    // compute and store cell gradients at once 
//    POLYREC polyRec(dcor);
//    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
//    CellData::Itr cell = cells.getItr(cellID);
//    polyRec.computeGradients(&states[cellID*PHYS::NBEQS], &centerNodes[cellID*PHYS::DIM], &kd, &cell);
//  }
//}
      
//////////////////////////////////////////////////////////////////////////////

//template <typename PHYS, typename POLYREC, typename LIMITER>
//__global__ void computeLimiterKernel(typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol,
//				     typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
//				     const CFuint nbCells,
//				     CFreal* states, 
//				     CFreal* nodes,
//				     CFreal* centerNodes,
//				     CFreal* ghostStates,
//				     CFreal* ghostNodes,
//				     CFreal* uX,
//				     CFreal* uY,
//				     CFreal* uZ,
//				     CFreal* limiter,
//				     CFreal* updateCoeff, 
//				     CFreal* rhs,
//				     CFreal* normals,
//				     CFint* isOutward,
//				     const CFuint* cellInfo,
//				     const CFuint* cellStencil,
//				     const CFuint* cellFaces,
//				     const CFuint* cellNodes,
//				     const CFint*  neighborTypes,
//				     const Framework::CellConn* cellConn)
//{    
//  // each thread takes care of computing the gradient for one single cell
//  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
//  
//  // __shared__ typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE> s_dcol[32];
//  // typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol = &s_dcol[threadIdx.x];
//  // dcol->init(gdcol);
//  
//  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
//  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
//  // dcor->init(gdcor);
//  
//  if (cellID < nbCells) {    
//    // compute all cell quadrature points at once (size of this array is overestimated)
//    CFreal midFaceCoord[PHYS::DIM*PHYS::DIM*2];
//    
//    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
//    CellData::Itr cell = cells.getItr(cellID);
//    const CFuint nbFacesInCell = cell.getNbFacesInCell();
//    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
//      computeFaceCentroid<PHYS>(&cell, f, nodes, &midFaceCoord[f*PHYS::DIM]);
//    }
//    
//    // compute cell-based limiter at once
//    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
//			   rhs, normals, uX, uY, uZ, isOutward);
//    LIMITER limt(dcol);
//    
//    if (dcor->currRes > dcor->limitRes && (dcor->limitIter > 0 && dcor->currIter < dcor->limitIter)) {	
//      limt.limit(&kd, &cell, &midFaceCoord[0], &limiter[cellID*PHYS::NBEQS]);
//    }
//    else {
//      if (!dcor->freezeLimiter) {
//	// historical modification of the limiter
//	CudaEnv::CFVec<CFreal,PHYS::NBEQS> tmpLimiter;
//	limt.limit(&kd, &cell, &midFaceCoord[0], &tmpLimiter[0]);
//	CFuint currID = cellID*PHYS::NBEQS;
//	for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar, ++currID) {
//	  limiter[currID] = min(tmpLimiter[iVar],limiter[currID]);
//	}
//      }
//    }
//  }
//}
  
//////////////////////////////////////////////////////////////////////////////

template <typename SCHEME, typename PHYS, typename PHYSNS, CFuint ORDER>
__global__ void computeStateLocalRHSKernel(typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof,
                                  typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
                                  typename PHYSNS::DTERM::template DeviceConfigOptions<NOTYPE>* dcopNS,
                                  const CFuint nbCells,
                                  const CFreal resFactor,
				  CFreal* states, 
                                  CFreal* gradients,
                                  CFreal* updateCoeff, 
				  CFreal* rhs,
                                  CFreal* solPntNormals,
                                  CFreal* flxPntNormals,
                                  CFint* faceDir,
                                  const CFuint nbSolPnts,
                                  const CFuint nbrFaces,
                                  const CFuint* faceFlxPntConn,
                                  const CFuint* stateIDs,
                                  const CFuint* neighbCellIDs,
                                  const CFuint* neighbFaceIDs,
                                  const CFuint* innerCellIsLeft,
                                  const CFuint nbrFlxPnts,
                                  const CFuint nbrSolSolDep,
                                  const CFuint* solSolDep,
                                  const CFuint nbrSolFlxDep,
                                  const CFuint* solFlxDep,
                                  const CFuint nbrFlxSolDep,
                                  const CFuint* flxSolDep,
                                  const CFreal* solPolyDerivAtSolPnts,
                                  const CFreal* solPolyValsAtFlxPnts,
                                  const CFuint* flxPntFlxDim,
                                  const CFreal* corrFctDiv,
                                  const CFreal* faceIntCoeff)
{    
  // one thread per cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (cellID < nbCells) 
  { 
    // current kernel data
    KernelData<CFreal> kd (nbCells, states, updateCoeff, rhs, solPntNormals, flxPntNormals, faceDir, nbSolPnts);

    // current flux data
    FluxData<typename SCHEME::MODEL,ORDER> currFd; 

    // initialize flux data
    currFd.initialize();
    
    // physical model
    typename SCHEME::MODEL pmodel(dcop);
    SCHEME fluxScheme(dcof);
    
    // current cell data
    CellData cells(nbCells, stateIDs, neighbCellIDs, neighbFaceIDs, innerCellIsLeft, nbrFaces, nbSolPnts, ORDER);
    
    // get current cell
    CellData::Itr cell = cells.getItr(cellID);
          
    const CFuint nbFlxPntFlx = SCHEME::MODEL::NBEQS*(ORDER+1)*2*PHYS::DIM;//8;
    
    const CFuint nbFaceFlxPntFlx = SCHEME::MODEL::NBEQS*(ORDER+1);
   
    const CFuint nbrFaceFlxPnts = (ORDER+1);

    const CFuint totNbrFlxPnts = (ORDER+1)*2*PHYS::DIM;

    const CFuint nbNormals = PHYS::DIM*PHYS::DIM;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx> flxPntFlx;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx> flxPntSol;
    
    flxPntFlx = 0.0;
    
    flxPntSol = 0.0;

    //CudaEnv::CFVecSlice<CFreal,nbrFaceFlxPnts> intCoeff(currFd.getFaceIntegrationCoef());

    //for (CFuint iFlxPnt = 0; iFlxPnt < nbrFaceFlxPnts; ++iFlxPnt)
    //{
      //intCoeff[iFlxPnt] = faceIntCoeff[iFlxPnt];
    //}
    //currFd.setFaceIntegrationCoef(iFlx,faceIntCoeff[iFlx]);
//if (cellID == 11) printf("iFlx , coeff\n");

 //     if (cellID == 1) printf("iFlx , coeff\n");
//if (cellID == 11) printf("hello %d\n", 0);
    // loop over sol pnts to compute flux
    for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
    {
      // get current state ID
      const CFuint stateID = cell.getStateID(iSolPnt);
      //printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
    //if (cellID == 0) printf("GPUstate: %f %f %f %f\n", kd.states[0], kd.states[1], kd.states[2], kd.states[3]);
    
      setFluxData(stateID, cellID, &kd, &currFd, iSolPnt);

      CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS*PHYS::DIM> grad(&gradients[stateID*SCHEME::MODEL::NBEQS*PHYS::DIM]);

      CudaEnv::CFVecSlice<CFreal,nbNormals> n(&(kd.solPntNormals[stateID*nbNormals]));

      CudaEnv::CFVecSlice<CFreal,nbNormals> nFd(currFd.getScaledNormal(iSolPnt));
      
      CFuint k = 0;

      for (CFuint i = 0; i < nbNormals; ++i) 
      {
        nFd[i] = n[i];
      }

      // get the flux
      fluxScheme.prepareComputation(&currFd, &pmodel);

      fluxScheme(&currFd, &pmodel, iSolPnt);

/// add diff fluxes here to currFD!
    }

    for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
    {
      // get current state ID
      const CFuint stateID = cell.getStateID(iSolPnt);

      setFluxData(stateID, cellID, &kd, &currFd, iSolPnt);

      // get current vector slice out of rhs
      CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> res(&rhs[stateID*SCHEME::MODEL::NBEQS]);

      // Loop over solution pnts to count the factor of all sol pnt polys
      for (CFuint jSolPnt = 0; jSolPnt < nbrSolSolDep; ++jSolPnt)
      { 
        const CFuint jSolIdx = solSolDep[iSolPnt*nbrSolSolDep+jSolPnt]; //(*m_solSolDep)[iSolPnt][jSolPnt];

        // Loop over deriv directions and sum them to compute divergence
        for (CFuint iDir = 0; iDir < PHYS::DIM; ++iDir)
        {
          const CFreal polyCoef = solPolyDerivAtSolPnts[iSolPnt*PHYS::DIM*nbSolPnts+iDir*nbSolPnts+jSolIdx];//(*m_solPolyDerivAtSolPnts)[jSolPnt][iDir][iSolIdx]; 
          
          // Loop over conservative fluxes 
          for (CFuint iEq = 0; iEq < SCHEME::MODEL::NBEQS; ++iEq)
          {
            // Store divFD in the vector that will be divFC
            res[iEq] -= polyCoef*(currFd.getFlux(jSolIdx, iDir)[iEq])*resFactor;

//if (cellID == 11 && abs(polyCoef*(currFd.getFlux(jSolIdx, iDir)[iEq])) > 1e-8) printf("State: %d, jSol: %d, iDir: %d, var: %d, up: %f, poly: %f, flx: %f\n",iSolPnt,jSolIdx,iDir,iEq,polyCoef*(currFd.getFlux(iSolPnt, iDir)[iEq]),polyCoef,currFd.getFlux(jSolIdx, iDir)[iEq]);  
	  }
        }
      }
    }

    CudaEnv::CFVec<CFreal,nbFlxPntFlx*nbNormals> flxPntGrads;

    // extrapolate the fluxes to the flux points
    for (CFuint iFlxPnt = 0; iFlxPnt < nbrFlxPnts; ++iFlxPnt)
    {
      const CFuint dim = flxPntFlxDim[iFlxPnt];

      // loop over sol pnts to compute flux
      for (CFuint iSolPnt = 0; iSolPnt < nbrFlxSolDep; ++iSolPnt)
      {
        const CFuint solIdx = flxSolDep[iFlxPnt*nbrFlxSolDep + iSolPnt];

        // Loop over conservative fluxes 
        for (CFuint iEq = 0; iEq < SCHEME::MODEL::NBEQS; ++iEq)
        {
          flxPntFlx[iFlxPnt*SCHEME::MODEL::NBEQS+iEq] += solPolyValsAtFlxPnts[iFlxPnt*nbSolPnts+solIdx]*currFd.getFlux(solIdx, dim)[iEq];

          flxPntSol[iFlxPnt*SCHEME::MODEL::NBEQS+iEq] += solPolyValsAtFlxPnts[iFlxPnt*nbSolPnts+solIdx]*states[cell.getStateID(solIdx)*SCHEME::MODEL::NBEQS+iEq];

          for (CFuint iDir = 0; iDir < PHYS::DIM; ++iDir)
          {

          }          
        }
      }
    }

    // set extrapolated states
    for (CFuint iState = 0; iState < nbrFlxPnts; ++iState)
    {
      for (CFuint iEq = 0; iEq < PHYS::NBEQS; ++iEq) 
      {
        currFd.getLstate(iState)[iEq] = flxPntSol[iState*PHYS::NBEQS+iEq];
      } 
    }

    for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
    {
      // get current state ID
      const CFuint stateID = cell.getStateID(iSolPnt);

      // get current vector slice out of rhs
      CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> res(&rhs[stateID*SCHEME::MODEL::NBEQS]);

      // add divhFD to the residual updates
      for (CFuint iFlxPnt = 0; iFlxPnt < nbrSolFlxDep; ++iFlxPnt)
      {
        const CFuint flxIdx = solFlxDep[iSolPnt*nbrSolFlxDep+iFlxPnt];

        // get the divergence of the correction function
        const CFreal divh = corrFctDiv[iSolPnt*nbrFlxPnts+flxIdx];

        // Fill in the corrections
        for (CFuint iVar = 0; iVar < SCHEME::MODEL::NBEQS; ++iVar)
        {
          res[iVar] += flxPntFlx[flxIdx*SCHEME::MODEL::NBEQS+iVar] * divh * resFactor;
//if (cellID==11 && abs(flxPntFlx[flxIdx*SCHEME::MODEL::NBEQS+iVar] * divh) > 1e-8) printf("State: %d, flx: %d, var: %d, update: %f\n",iSolPnt,flxIdx,iVar,flxPntFlx[flxIdx*SCHEME::MODEL::NBEQS+iVar] * divh);  
        }
      }
    }

    // reset flx pnt fluxes  
    flxPntFlx = 0.0;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx> flxPntSolNeighb;
    
    for (CFuint iFlxPnt = 0; iFlxPnt < nbFlxPntFlx; ++iFlxPnt) {flxPntSolNeighb[iFlxPnt] = 0.0;}

    // current neighb cell data
    CellData cells2(nbCells, stateIDs, neighbCellIDs, neighbFaceIDs, innerCellIsLeft, nbrFaces, nbSolPnts, ORDER);

    for (CFuint iFace = 0; iFace < nbrFaces; ++iFace)
    {
      const CFuint neighbCellID = cell.getNeighbCellID(iFace);  

      // get current cell
      CellData::Itr cell2 = cells2.getItr(neighbCellID);

      CFuint jFaceIdx = 100;

      for (CFuint jFace = 0; jFace < nbrFaces; ++jFace)
      {
        if (cell2.getNeighbCellID(jFace) == cellID)
        {
          jFaceIdx = jFace; 
          break;
        }
      }

      CFreal waveSpeedUpd = 0.0;

      if (jFaceIdx != 100)
      {
        const CFuint faceID = cell.getNeighbFaceID(iFace);

        const bool isLEFT = (bool) cell.getInnerCellIsLeft(iFace);

      // loop over face flx pnts
      for (CFuint iFlxPnt = 0; iFlxPnt < nbrFaceFlxPnts; ++iFlxPnt)
      { 
        // @TODO check if this also works for non QUADs
        const CFuint flxIdx = faceFlxPntConn[iFace*nbrFaceFlxPnts+iFlxPnt];
        const CFuint jFlxIdx = faceFlxPntConn[jFaceIdx*nbrFaceFlxPnts+nbrFaceFlxPnts-1-iFlxPnt];

        // loop over sol pnts to compute sol at flx pnt
        for (CFuint iSolPnt = 0; iSolPnt < nbrFlxSolDep; ++iSolPnt)
        {
          const CFuint solIdx = flxSolDep[jFlxIdx*nbrFlxSolDep+iSolPnt]; 

          // Loop over conservative vars 
          for (CFuint iEq = 0; iEq < SCHEME::MODEL::NBEQS; ++iEq)
          {
            flxPntSolNeighb[flxIdx*SCHEME::MODEL::NBEQS+iEq] += solPolyValsAtFlxPnts[jFlxIdx*nbSolPnts+solIdx]*states[cell2.getStateID(solIdx)*SCHEME::MODEL::NBEQS+iEq];
          }
        }

        for (CFuint iEq = 0; iEq < PHYS::NBEQS; ++iEq) 
        {
          currFd.getRstate(flxIdx)[iEq] = flxPntSolNeighb[flxIdx*PHYS::NBEQS+iEq];
        } 

        CudaEnv::CFVecSlice<CFreal,PHYS::DIM> n(&(kd.flxPntNormals[faceID*nbrFaceFlxPnts*PHYS::DIM+iFlxPnt*PHYS::DIM]));

        CudaEnv::CFVecSlice<CFreal,PHYS::DIM> nFd(currFd.getFlxScaledNormal(flxIdx));

        for (CFuint i = 0; i < PHYS::DIM; ++i) 
        {
          nFd[i] = n[i];
        }

        // get the flux
        fluxScheme.prepareComputation(&currFd, &pmodel);

        fluxScheme(&currFd, &pmodel, iFlxPnt, flxIdx, faceIntCoeff[iFlxPnt], isLEFT, waveSpeedUpd);

        // extrapolate the fluxes to the flux points
        for (CFuint iSolPnt = 0; iSolPnt < nbrFlxSolDep; ++iSolPnt)
        {     
          const CFuint solIdx = flxSolDep[flxIdx*nbrFlxSolDep+iSolPnt];

          // get current state ID
          const CFuint stateID = cell.getStateID(solIdx);

          // get current vector slice out of rhs
          CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> res(&rhs[stateID*SCHEME::MODEL::NBEQS]);   
          
          // divergence of the correction function
          const CFreal divh = corrFctDiv[solIdx*nbrFlxPnts+flxIdx] * faceDir[cellID*totNbrFlxPnts+flxIdx];

          // Fill in the corrections
          for (CFuint iVar = 0; iVar < SCHEME::MODEL::NBEQS; ++iVar)
          {
            res[iVar] -= currFd.getInterfaceFlux(flxIdx)[iVar] * divh * resFactor;
//if (cellID==768) printf("resID: %d, State: %d, flx: %d, var: %d, updateFace: %f, flx %f, divh %f\n",stateID*SCHEME::MODEL::NBEQS+iVar,solIdx,flxIdx,iVar,-currFd.getInterfaceFlux(flxIdx)[iVar] * divh, currFd.getInterfaceFlux(flxIdx)[iVar],divh); 
          }
        }
      }

      //CFreal* waveSpeedUpd = currFd.getUpdateCoeff();

      for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
      {  
        // get current state ID
        const CFuint stateID = cell.getStateID(iSolPnt);

        updateCoeff[stateID] += waveSpeedUpd;
      }
 
      //currFd.resetUpdateCoeff();
      }
    }
  }
}
  
//////////////////////////////////////////////////////////////////////////////

template <typename SCHEME, typename PHYS, typename PHYSNS, CFuint ORDER>
__global__ void computeGradientsKernel(typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
                                       typename PHYSNS::DTERM::template DeviceConfigOptions<NOTYPE>* dcopNS,
                                       typename PHYSNS::PTERM::template DeviceConfigOptions<NOTYPE>* dcopNSConv,
                                       const CFuint nbCells,
				       CFreal* states, 
                                       CFreal* gradients,
                                       CFreal* solPntNormals,
                                       CFreal* flxPntNormals,
                                       CFint* faceDir,
                                       const CFuint nbSolPnts,
                                       const CFuint nbrFaces,
                                       const CFuint* faceFlxPntConn,
                                       const CFuint* stateIDs,
                                       const CFuint* neighbCellIDs,
                                       const CFuint* neighbFaceIDs,
                                       const CFuint* innerCellIsLeft,
                                       const CFuint nbrFlxPnts,
                                       const CFuint nbrSolSolDep,
                                       const CFuint* solSolDep,
                                       const CFuint nbrSolFlxDep,
                                       const CFuint* solFlxDep,
                                       const CFuint nbrFlxSolDep,
                                       const CFuint* flxSolDep,
                                       const CFreal* solPolyDerivAtSolPnts,
                                       const CFreal* solPolyValsAtFlxPnts,
                                       const CFuint* flxPntFlxDim,
                                       const CFreal* corrFctDiv)
{    
  // one thread per cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (cellID < nbCells) 
  { 
    // current kernel data
    //KernelData<CFreal> kd (nbCells, states, updateCoeff, rhs, solPntNormals, flxPntNormals, faceDir, nbSolPnts);
    
    // physical model
    typename SCHEME::MODEL pmodel(dcop);
    //typename PHYSNS pmodelNS(dcopNS);
    PHYSNS pmodelNS(dcopNS,dcopNSConv);    

    // current cell data
    CellData cells(nbCells, stateIDs, neighbCellIDs, neighbFaceIDs, innerCellIsLeft, nbrFaces, nbSolPnts, ORDER);
    
    // get current cell
    CellData::Itr cell = cells.getItr(cellID);
          
    const CFuint nbFlxPntFlx = SCHEME::MODEL::NBEQS*(ORDER+1)*2*PHYS::DIM;//8;
    
    const CFuint nbFaceFlxPntFlx = SCHEME::MODEL::NBEQS*(ORDER+1);
   
    const CFuint nbrFaceFlxPnts = (ORDER+1);

    const CFuint totNbrFlxPnts = (ORDER+1)*2*PHYS::DIM;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx> flxPntFlx;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx> flxPntSol;

    CudaEnv::CFVec<CFreal,SCHEME::MODEL::NBEQS> stateGradVars;
    //typename MathTypes<CFreal, GPU, SCHEME::MODEL::NBEQS>::VEC stateGradVars;
    
    flxPntFlx = 0.0;
    
    flxPntSol = 0.0;

    stateGradVars = 0.0;

    // loop over sol pnts to compute flux
    for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
    {
      // get current state ID
      const CFuint stateID = cell.getStateID(iSolPnt);

      //typename MathTypes<CFreal, GPU, SCHEME::MODEL::NBEQS>::SLICEVEC currState(&states[stateID*SCHEME::MODEL::NBEQS]);

      CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> currState(&states[stateID*SCHEME::MODEL::NBEQS]);

      pmodelNS.getUpdateVS()->setGradientVars(&currState[0],&stateGradVars[0]);

      CudaEnv::CFVecSlice<CFreal,PHYS::DIM*PHYS::DIM> currNormals(&solPntNormals[stateID*PHYS::DIM*PHYS::DIM]);

      // get current vector slice out of gradients
      CudaEnv::CFVecSlice<CFreal,PHYS::DIM*SCHEME::MODEL::NBEQS> grad(&gradients[stateID*SCHEME::MODEL::NBEQS*PHYS::DIM]);

      // Loop over solution pnts to count the factor of all sol pnt polys
      for (CFuint jSolPnt = 0; jSolPnt < nbrSolSolDep; ++jSolPnt)
      { 
        const CFuint jSolIdx = solSolDep[iSolPnt*nbrSolSolDep+jSolPnt]; //(*m_solSolDep)[iSolPnt][jSolPnt];

        // Loop over deriv directions and sum them to compute divergence
        for (CFuint iDir = 0; iDir < PHYS::DIM; ++iDir)
        {
          const CFreal polyCoef = solPolyDerivAtSolPnts[iSolPnt*PHYS::DIM*nbSolPnts+iDir*nbSolPnts+jSolIdx];

          for (CFuint jDir = 0; jDir < PHYS::DIM; ++jDir)
          {
            // Loop over conservative fluxes 
            for (CFuint iEq = 0; iEq < SCHEME::MODEL::NBEQS; ++iEq)
            {
              //if (cellID == 11) printf("iSol: %d, iEq: %d, iDir: %d: %f\n", iSolPnt, iEq, jDir, polyCoef*currNormals[iDir*PHYS::DIM+jDir]*stateGradVars[iEq]); 

              // Store divFD in the vector that will be divFC
              grad[iEq*PHYS::DIM+jDir] += polyCoef*currNormals[iDir*PHYS::DIM+jDir]*stateGradVars[iEq];//*states[stateID*SCHEME::MODEL::NBEQS+iEq]; 

              //if (cellID == 11) printf("after  iSol: %d, iEq: %d, iDir: %d: %f\n", iSolPnt, iEq, jDir, grad[iEq*PHYS::DIM+jDir]); 
	    }
          }
        }
      }
    }

    // extrapolate the fluxes to the flux points
    for (CFuint iFlxPnt = 0; iFlxPnt < nbrFlxPnts; ++iFlxPnt)
    {
      const CFuint dim = flxPntFlxDim[iFlxPnt];

      // loop over sol pnts to compute flux
      for (CFuint iSolPnt = 0; iSolPnt < nbrFlxSolDep; ++iSolPnt)
      {
        const CFuint solIdx = flxSolDep[iFlxPnt*nbrFlxSolDep + iSolPnt];

        CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> currState(&states[cell.getStateID(solIdx)*SCHEME::MODEL::NBEQS]);

        pmodelNS.getUpdateVS()->setGradientVars(&currState[0],&stateGradVars[0]);

        // Loop over conservative fluxes 
        for (CFuint iEq = 0; iEq < SCHEME::MODEL::NBEQS; ++iEq)
        {
          flxPntSol[iFlxPnt*SCHEME::MODEL::NBEQS+iEq] += solPolyValsAtFlxPnts[iFlxPnt*nbSolPnts+solIdx]*stateGradVars[iEq];          
        }
      }
    }

    // reset flx pnt fluxes  
    flxPntFlx = 0.0;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx> flxPntSolNeighb;
    
    for (CFuint iFlxPnt = 0; iFlxPnt < nbFlxPntFlx; ++iFlxPnt) {flxPntSolNeighb[iFlxPnt] = 0.0;}

    // current neighb cell data
    CellData cells2(nbCells, stateIDs, neighbCellIDs, neighbFaceIDs, innerCellIsLeft, nbrFaces, nbSolPnts, ORDER);

    for (CFuint iFace = 0; iFace < nbrFaces; ++iFace)
    {
      const CFuint neighbCellID = cell.getNeighbCellID(iFace);  

      // get current cell
      CellData::Itr cell2 = cells2.getItr(neighbCellID);

      CFuint jFaceIdx = 100;

      for (CFuint jFace = 0; jFace < nbrFaces; ++jFace)
      {
        if (cell2.getNeighbCellID(jFace) == cellID)
        {
          jFaceIdx = jFace; 
          break;
        }
      }

      if (jFaceIdx != 100)
      {
        const CFuint faceID = cell.getNeighbFaceID(iFace);

        const bool isLEFT = (bool) cell.getInnerCellIsLeft(iFace);

        // loop over face flx pnts
        for (CFuint iFlxPnt = 0; iFlxPnt < nbrFaceFlxPnts; ++iFlxPnt)
        { 
          // @TODO check if this also works for non QUADs
          const CFuint flxIdx = faceFlxPntConn[iFace*nbrFaceFlxPnts+iFlxPnt];
          const CFuint jFlxIdx = faceFlxPntConn[jFaceIdx*nbrFaceFlxPnts+nbrFaceFlxPnts-1-iFlxPnt];

          const CFreal dirFactor = faceDir[cellID*totNbrFlxPnts+flxIdx];

          // loop over sol pnts to compute sol at flx pnt
          for (CFuint iSolPnt = 0; iSolPnt < nbrFlxSolDep; ++iSolPnt)
          {
            const CFuint solIdx = flxSolDep[jFlxIdx*nbrFlxSolDep+iSolPnt]; 

            CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> currState(&states[cell2.getStateID(solIdx)*SCHEME::MODEL::NBEQS]);

            pmodelNS.getUpdateVS()->setGradientVars(&currState[0],&stateGradVars[0]);

            // Loop over conservative vars 
            for (CFuint iEq = 0; iEq < SCHEME::MODEL::NBEQS; ++iEq)
            {
              flxPntSolNeighb[flxIdx*SCHEME::MODEL::NBEQS+iEq] += solPolyValsAtFlxPnts[jFlxIdx*nbSolPnts+solIdx]*stateGradVars[iEq];
            }
          }

          // extrapolate the fluxes to the flux points
          for (CFuint iSolPnt = 0; iSolPnt < nbrFlxSolDep; ++iSolPnt)
          {     
            const CFuint solIdx = flxSolDep[flxIdx*nbrFlxSolDep+iSolPnt];

            // get current state ID
            const CFuint stateID = cell.getStateID(solIdx); 

            // get current vector slice out of gradients
            CudaEnv::CFVecSlice<CFreal,PHYS::DIM*SCHEME::MODEL::NBEQS> grad(&gradients[stateID*SCHEME::MODEL::NBEQS*PHYS::DIM]);

            // divergence of the correction function
            const CFreal divh = corrFctDiv[solIdx*nbrFlxPnts+flxIdx];
           
            for (CFuint iEq = 0; iEq < SCHEME::MODEL::NBEQS; ++iEq)
            {
              const CFreal corrFactor = 0.5*(flxPntSolNeighb[flxIdx*SCHEME::MODEL::NBEQS+iEq]-flxPntSol[flxIdx*SCHEME::MODEL::NBEQS+iEq]);

              // Loop over deriv directions and sum them to compute divergence
              for (CFuint iDir = 0; iDir < PHYS::DIM; ++iDir)
              {
                for (CFuint jDir = 0; jDir < PHYS::DIM; ++jDir)
                {
                  // Store divFD in the vector that will be divFC
                  grad[iEq*PHYS::DIM+jDir] += divh*corrFactor*flxPntNormals[faceID*nbrFaceFlxPnts*PHYS::DIM+iFlxPnt*PHYS::DIM+jDir]*dirFactor; 

              //if (cellID == 11) printf("iSol: %d, iEq: %d, iFlx %d, iDir: %d: %e\n", iSolPnt, iEq, flxIdx, jDir, divh*corrFactor*flxPntNormals[faceID*nbrFaceFlxPnts*PHYS::DIM+iFlxPnt*PHYS::DIM+jDir]*dirFactor);  
	        }
              }
            }
          }
        }
      }
    }

    for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
    {  
      // get current state ID
      const CFuint stateID = cell.getStateID(iSolPnt);

      // get current vector slice out of gradients
      CudaEnv::CFVecSlice<CFreal,PHYS::DIM*SCHEME::MODEL::NBEQS> grad(&gradients[stateID*SCHEME::MODEL::NBEQS*PHYS::DIM]);

      // Loop over deriv directions and sum them to compute divergence
      for (CFuint iDir = 0; iDir < PHYS::DIM; ++iDir)
      {
        CFreal nJacob2 = 0.0;

        for (CFuint jDir = 0; jDir < PHYS::DIM; ++jDir)
        {
          nJacob2 += solPntNormals[stateID*PHYS::DIM*PHYS::DIM+iDir*PHYS::DIM+jDir]*solPntNormals[stateID*PHYS::DIM*PHYS::DIM+iDir*PHYS::DIM+jDir];
        }

        const CFreal invJacob = 1/pow(nJacob2,0.5);

        for (CFuint jDir = 0; jDir < PHYS::DIM; ++jDir)
        {
          // Loop over conservative fluxes 
          for (CFuint iEq = 0; iEq < SCHEME::MODEL::NBEQS; ++iEq)
          {
            // Store divFD in the vector that will be divFC
            grad[iEq*PHYS::DIM+jDir] *= invJacob;  
	  }
        }
      }
        //if (cellID == 11) printf("iSol: %d, invJacob: %e\n", iSolPnt, temp);
    }
  }
}

//////////////////////////////////////////////////////////////////////////////

template <typename SCHEME, typename PHYSICS, typename PHYSICSNS, CFuint ORDER, CFuint NB_BLOCK_THREADS>
void ConvDiffRHSFluxReconstructionCUDA<SCHEME,PHYSICS,PHYSICSNS,ORDER,NB_BLOCK_THREADS>::execute()
{
  using namespace COOLFluiD::Framework;
  using namespace COOLFluiD::Common;
  
  CFTRACEBEGIN;
  
  CFLog(VERBOSE, "ConvDiffRHSFluxReconstructionCUDA::execute() START\n");
  
  // get the elementTypeData
  SafePtr< vector<ElementTypeData> > elemType = MeshDataStack::getActive()->getElementTypeData();

  // get InnerCells TopologicalRegionSet
  SafePtr<TopologicalRegionSet> cells = MeshDataStack::getActive()->getTrs("InnerCells");

  // get the geodata of the geometric entity builder and set the TRS
  StdTrsGeoBuilder::GeoData& geoDataCell = m_cellBuilder->getDataGE();
  geoDataCell.trs = cells;
  
  // get InnerFaces TopologicalRegionSet
  SafePtr<TopologicalRegionSet> faces = MeshDataStack::getActive()->getTrs("InnerFaces");

  // get the face start indexes
  vector< CFuint >& innerFacesStartIdxs = getMethodData().getInnerFacesStartIdxs();

  // get number of face orientations
  const CFuint nbrFaceOrients = innerFacesStartIdxs.size()-1;

  // get the geodata of the face builder and set the TRSs
  FaceToCellGEBuilder::GeoData& geoDataFace = m_faceBuilder->getDataGE();
  geoDataFace.cellsTRS = cells;
  geoDataFace.facesTRS = faces;
  geoDataFace.isBoundary = false;
  
  // loop over element types, for the moment there should only be one
  const CFuint nbrElemTypes = elemType->size();
  cf_assert(nbrElemTypes == 1);
  
  // get start and end indexes for this type of element
  cf_assert((*elemType)[0].getStartIdx() == 0);
  const CFuint nbCells   = (*elemType)[0].getEndIdx();
  cf_assert(nbCells > 0);
  
  //initializeComputationRHS();

  const CFuint nbStates = socket_states.getDataHandle().size();
  cf_assert(nbStates > 0);

  CFLog(VERBOSE, "nbCells: " << nbCells << ", nbStates: " << nbStates << "\n");

  DataHandle<CFreal> updateCoeff = socket_updateCoeff.getDataHandle();
  DataHandle<CFreal> rhs = socket_rhs.getDataHandle(); 
  DataHandle<CFreal> solPntNormals = socket_solPntNormals.getDataHandle(); 
  DataHandle<CFreal> flxPntNormals = socket_flxPntNormals.getDataHandle(); 
  DataHandle<CFint> faceDir = socket_faceDir.getDataHandle(); 
  DataHandle<CFreal> gradients = socket_gradientsCUDA.getDataHandle();
 

  SafePtr<SCHEME> lf  = getMethodData().getRiemannFlux().d_castTo<SCHEME>();
  SafePtr<typename PHYSICS::PTERM> phys = PhysicalModelStack::getActive()->getImplementor()->
    getConvectiveTerm().d_castTo<typename PHYSICS::PTERM>();

  SafePtr<typename PHYSICSNS::DTERM> physNS = PhysicalModelStack::getActive()->getImplementor()->
    getDiffusiveTerm().d_castTo<typename PHYSICSNS::DTERM>();

  SafePtr<typename PHYSICSNS::PTERM> physNSConv = PhysicalModelStack::getActive()->getImplementor()->
    getConvectiveTerm().d_castTo<typename PHYSICSNS::PTERM>();
  
#ifdef CF_HAVE_CUDA
  typedef typename SCHEME::template DeviceFunc<GPU, PHYSICS, ORDER> FluxScheme;  
#else
  typedef typename SCHEME::template DeviceFunc<CPU, PHYSICS, ORDER> FluxScheme;
#endif 
  
  if (m_onGPU) 
  {
#ifdef CF_HAVE_CUDA

    //CudaEnv::CudaTimer& timer = CudaEnv::CudaTimer::getInstance();
    //timer.start();
    
    // copy of data that change at every iteration
    socket_states.getDataHandle().getGlobalArray()->put(); 
    socket_gradientsCUDA.getDataHandle().getLocalArray()->put();
    socket_rhs.getDataHandle().getLocalArray()->put(); 
    socket_updateCoeff.getDataHandle().getLocalArray()->put();

    
    //CFLog(VERBOSE, "nb normals: " << socket_solPntNormals.getDataHandle().size() << ", n0: " << socket_solPntNormals.getDataHandle()[0] << "\n");

    socket_faceDir.getDataHandle().getLocalArray()->put();
    socket_solPntNormals.getDataHandle().getLocalArray()->put();
    socket_flxPntNormals.getDataHandle().getLocalArray()->put();


    DataHandle<Framework::State*, Framework::GLOBAL > statesI = socket_states.getDataHandle();
     
    //CFLog(VERBOSE, "ConvDiffRHSFluxReconstructionCUDA::execute() => CPU-->GPU data transfer took " << timer.elapsed() << " s\n");
    //timer.start();
    
    ConfigOptionPtr<SCHEME,  NOTYPE, GPU> dcof(lf);
    ConfigOptionPtr<typename PHYSICS::PTERM, NOTYPE, GPU> dcop(phys);
    ConfigOptionPtr<typename PHYSICSNS::DTERM, NOTYPE, GPU> dcopNS(physNS);
    ConfigOptionPtr<typename PHYSICSNS::PTERM, NOTYPE, GPU> dcopNSConv(physNSConv);

    const CFuint blocksPerGrid = CudaEnv::CudaDeviceManager::getInstance().getBlocksPerGrid(nbCells);
    const CFuint nThreads = CudaEnv::CudaDeviceManager::getInstance().getNThreads();
    CFLog(VERBOSE, "blocksPerGrid: " << blocksPerGrid << ", threads: " << nThreads << "\n");

    // boolean telling whether there is a diffusive term
    const bool hasDiffTerm = getMethodData().hasDiffTerm() || getMethodData().hasArtificialViscosity();

//CFuint megabytesToUse = 24;
//size_t newHeapSize = 1024 * 1000 * megabytesToUse;
//hipDeviceSetLimit(hipLimitMallocHeapSize, newHeapSize);
//printf("Adjusted heap size to be %d\n",(int) newHeapSize);

    //dim3 blocks(m_nbBlocksPerGridX, m_nbBlocksPerGridY);
    
    //hipFuncSetCacheConfig(reinterpret_cast<const void*>("computeGradientsKernel"), hipFuncCachePreferL1);

    // get residual factor
    const CFreal resFactor = getMethodData().getResFactor();
    
    // hipFuncSetCacheConfig(reinterpret_cast<const void*>("computeFluxKernel"), hipFuncCachePreferL1);

    // if there is a diffusive term, compute the gradients
    if (hasDiffTerm)
    {
      computeGradientsKernel<FluxScheme,PHYSICS,PHYSICSNS,ORDER> <<<blocksPerGrid,nThreads>>>(
                                       dcop.getPtr(),
                                       dcopNS.getPtr(),
                                       dcopNSConv.getPtr(),
                                       nbCells,
				       socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
                                       gradients.getLocalArray()->ptrDev(), 
                                       solPntNormals.getLocalArray()->ptrDev(),
                                       flxPntNormals.getLocalArray()->ptrDev(),
                                       faceDir.getLocalArray()->ptrDev(),
                                       m_nbrSolPnts,
                                       4,
                                       m_faceFlxPntConn2.ptrDev(),
                                       m_stateIDs.ptrDev(),
                                       m_neighbCellIDs.ptrDev(),
                                       m_neighbFaceIDs.ptrDev(),
                                       m_innerCellIsLeft.ptrDev(),
                                       m_nbrFlxPnts,
                                       m_nbrSolSolDep,
                                       m_solSolDep2.ptrDev(),
                                       m_nbrFlxDep,
                                       m_solFlxDep2.ptrDev(),
                                       m_nbrSolDep,
                                       m_flxSolDep2.ptrDev(),
                                       m_solPolyDerivAtSolPnts2.ptrDev(),
                                       m_solPolyValsAtFlxPnts2.ptrDev(),
                                       m_flxPntFlxDim2.ptrDev(),
                                       m_corrFctDiv2.ptrDev());
    }

    // compute the convective flux in each cell
    computeStateLocalRHSKernel<FluxScheme,PHYSICS,PHYSICSNS,ORDER> <<<blocksPerGrid,nThreads>>> 
      (dcof.getPtr(),
       dcop.getPtr(),
       dcopNS.getPtr(),
       nbCells,
       resFactor,
       socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
       gradients.getLocalArray()->ptrDev(), 
       updateCoeff.getLocalArray()->ptrDev(), 
       rhs.getLocalArray()->ptrDev(),
       solPntNormals.getLocalArray()->ptrDev(),
       flxPntNormals.getLocalArray()->ptrDev(),
       faceDir.getLocalArray()->ptrDev(),
       m_nbrSolPnts,
       4,
       m_faceFlxPntConn2.ptrDev(),
       m_stateIDs.ptrDev(),
       m_neighbCellIDs.ptrDev(),
       m_neighbFaceIDs.ptrDev(),
       m_innerCellIsLeft.ptrDev(),
       m_nbrFlxPnts,
       m_nbrSolSolDep,
       m_solSolDep2.ptrDev(),
       m_nbrFlxDep,
       m_solFlxDep2.ptrDev(),
       m_nbrSolDep,
       m_flxSolDep2.ptrDev(),
       m_solPolyDerivAtSolPnts2.ptrDev(),
       m_solPolyValsAtFlxPnts2.ptrDev(),
       m_flxPntFlxDim2.ptrDev(),
       m_corrFctDiv2.ptrDev(),
       m_faceIntegrationCoefs2.ptrDev());
   
    hipDeviceSynchronize();
    
    //for (CFuint i = 0; i < m_solPolyValsAtFlxPnts2.size(); ++i) {CFLog(INFO, "thing: " << m_solPolyValsAtFlxPnts2[i] << "\n");}
    
    //CFLog(INFO, "After Kernel, size: " << socket_states.getDataHandle().size() << "\n");
    
    //CFLog(VERBOSE, "ConvDiffRHSFluxReconstructionCUDA::execute() => computeFluxKernel took " << timer.elapsed() << " s\n");
    
    //for (CFuint i = 0; i < rhs.size(); ++i) {CFLog(INFO, "res before: " << rhs[i] << "\n");}
    
    //RealVector rhsB;
    //rhsB.resize(rhs.size());
    //for (CFuint i = 0; i < rhs.size(); ++i) {rhsB[i] = rhs[i];}
    
    //timer.start();
    rhs.getLocalArray()->get();
    updateCoeff.getLocalArray()->get();
    gradients.getLocalArray()->get();
    
    //for (CFuint i = 0; i < rhs.size(); ++i) {CFLog(INFO, "res after: " << rhs[i]-rhsB[i] << "\n");}
    //CFLog(VERBOSE, "ConvDiffRHSFluxReconstructionCUDA::execute() => GPU-->CPU data transfer took " << timer.elapsed() << " s\n");
    //CFLog(INFO, "resSize: " << rhs.size() << "\n");
    //for (CFuint i = 0; i < rhs.size(); ++i)
    //{
      //if (abs(rhs[i]) > 1.0e-10) CFLog(INFO, "res " << i << ": " << rhs[i] << "\n");
    //}

  #endif
  }
  else 
  {
  // boolean telling whether there is a diffusive term
  const bool hasDiffTerm = getMethodData().hasDiffTerm() || getMethodData().hasArtificialViscosity();

  // loop over element types, for the moment there should only be one
  for (m_iElemType = 0; m_iElemType < nbrElemTypes; ++m_iElemType)
  {
    // get start and end indexes for this type of element
    const CFuint startIdx = (*elemType)[m_iElemType].getStartIdx();
    const CFuint endIdx   = (*elemType)[m_iElemType].getEndIdx();

    // loop over cells
    for (CFuint elemIdx = startIdx; elemIdx < endIdx; ++elemIdx)
    {
      // build the GeometricEntity
      geoDataCell.idx = elemIdx;
      m_cell = m_cellBuilder->buildGE();

      // get the states in this cell
      m_cellStates = m_cell->getStates();
      
      // if the states in the cell are parallel updatable or the gradients need to be computed, set the cell data
      if ((*m_cellStates)[0]->isParUpdatable() || hasDiffTerm)
      {
	// set the cell data
	setCellData();
      }
      
      // if the states in the cell are parallel updatable, compute the divergence of the discontinuous flx (-divFD+divhFD)
      if ((*m_cellStates)[0]->isParUpdatable())
      {
	// compute the divergence of the discontinuous flux (-divFD+divhFD)
	computeDivDiscontFlx(m_divContFlx);
      
	// update RHS
        updateRHS();
      } 
      
      // if there is a diffusive term, compute the gradients
      if (hasDiffTerm)
      {
	computeGradients();
      }
      
      // print out the residual updates for debugging
      if(m_cell->getID() == 35) //true) //
      {
	CFLog(VERBOSE, "ID  = " << (*m_cellStates)[0]->getLocalID() << "\n");
        CFLog(VERBOSE, "coords  = " << (*m_cellStates)[0]->getCoordinates() << "\n");
        CFLog(VERBOSE, "UpdateTotal = \n");
        // get the datahandle of the rhs
        DataHandle< CFreal > rhs = socket_rhs.getDataHandle();
        for (CFuint iState = 0; iState < m_nbrSolPnts; ++iState)
        {
          CFuint resID = m_nbrEqs*( (*m_cellStates)[iState]->getLocalID() );
          for (CFuint iVar = 0; iVar < m_nbrEqs; ++iVar)
          {
            CFLog(VERBOSE, "" << rhs[resID+iVar] << " ");
          }
          CFLog(VERBOSE,"\n");
          DataHandle<CFreal> updateCoeff = socket_updateCoeff.getDataHandle();
          CFLog(VERBOSE, "UpdateCoeff: " << updateCoeff[(*m_cellStates)[iState]->getLocalID()] << "\n");
	  CFLog(VERBOSE, "state " << iState << ": " << *(((*m_cellStates)[iState])->getData()) << "\n");
        }
      }
      
      if(m_cell->getID() == 35 && hasDiffTerm)
      {
	// get the gradients
        DataHandle< vector< RealVector > > gradients = socket_gradients.getDataHandle();

        for (CFuint iState = 0; iState < m_nbrSolPnts; ++iState)
        {
	  CFuint solID = ((*m_cellStates)[iState])->getLocalID();
          for (CFuint iGrad = 0; iGrad < m_nbrEqs; ++iGrad)
          {
	    CFLog(VERBOSE, "total gradient " << iGrad << " of  " << iState << ": " << gradients[solID][iGrad] << "\n");
          } 
        }
        for (CFuint iState = 0; iState < m_nbrSolPnts; ++iState)
        {
	  CFLog(VERBOSE, "state " << iState << ": " << *(((*m_cellStates)[iState])->getData()) << "\n");
	}
      }
      
      //release the GeometricEntity
      m_cellBuilder->releaseGE();
    }
  }
  
  //// Loop over faces to calculate fluxes and interface fluxes in the flux points
  
  // loop over different orientations
  for (m_orient = 0; m_orient < nbrFaceOrients; ++m_orient)
  {
    CFLog(VERBOSE, "Orient = " << m_orient << "\n");
    // start and stop index of the faces with this orientation
    const CFuint faceStartIdx = innerFacesStartIdxs[m_orient  ];
    const CFuint faceStopIdx  = innerFacesStartIdxs[m_orient+1];

    // loop over faces with this orientation
    for (CFuint faceID = faceStartIdx; faceID < faceStopIdx; ++faceID)
    {
      // build the face GeometricEntity
      geoDataFace.idx = faceID;
      m_face = m_faceBuilder->buildGE();

      // get the neighbouring cells
      m_cells[LEFT ] = m_face->getNeighborGeo(LEFT );
      m_cells[RIGHT] = m_face->getNeighborGeo(RIGHT);

      // get the states in the neighbouring cells
      m_states[LEFT ] = m_cells[LEFT ]->getStates();
      m_states[RIGHT] = m_cells[RIGHT]->getStates();

      // if one of the neighbouring cells is parallel updatable or if the gradients have to be computed, set the bnd face data
      if ((*m_states[LEFT ])[0]->isParUpdatable() || (*m_states[RIGHT])[0]->isParUpdatable() || hasDiffTerm)
      {
	// set the bnd face data
        setFaceData(m_face->getID());//faceID

	// compute the states in the flx pnts
        computeFlxPntStates();

	// compute the interface flux
	computeInterfaceFlxCorrection();
          
	// compute the wave speed updates
        computeWaveSpeedUpdates(m_waveSpeedUpd);

        // update the wave speed
        updateWaveSpeed();
      }
	
	// if one of the neighbouring cells is parallel updatable, compute the correction flux
      if ((*m_states[LEFT ])[0]->isParUpdatable() || (*m_states[RIGHT])[0]->isParUpdatable())
      {
	// compute the correction for the left neighbour
	computeCorrection(LEFT, m_divContFlxL);
	
	// compute the correction for the right neighbour
	computeCorrection(RIGHT, m_divContFlxR);
	
	// update RHS
	updateRHSBothSides();
      }
      
      // if there is a diffusive term, compute the gradients
      if (hasDiffTerm)
      {
	// compute the face correction term of the corrected gradients
        computeGradientFaceCorrections();
      }

      // release the GeometricEntity
      m_faceBuilder->releaseGE();
    }
  }

    //DataHandle< CFreal > rhs = socket_rhs.getDataHandle();

    //for (CFuint i = 0; i < rhs.size(); ++i)
    //{
      //if (abs(rhs[i]) > 1.0e-10) CFLog(INFO, "res " << i << ": " << rhs[i] << "\n");
    //}
  }
  
  //finalizeComputationRHS();
  
  CFLog(VERBOSE, "ConvDiffRHSFluxReconstructionCUDA::execute() END\n");
  
  CFTRACEEND;
}

//////////////////////////////////////////////////////////////////////////////

    } // namespace FluxReconstructionMethod

} // namespace COOLFluiD
