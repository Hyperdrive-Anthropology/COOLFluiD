#include "hip/hip_runtime.h"
#include "FluxReconstructionCUDA/ConvRHSFluxReconstructionCUDA.hh"
#include "Framework/MeshData.hh"
#include "Framework/CellConn.hh"
#include "Config/ConfigOptionPtr.hh"
#include "Framework/CudaDeviceManager.hh"
#include "Common/CUDA/CFVec.hh"
#include "Framework/CudaTimer.hh"

#include "FluxReconstructionMethod/FluxData.hh"
#include "FluxReconstructionMethod/KernelData.hh"
#include "FluxReconstructionMethod/CellData.hh"

#include "FluxReconstructionCUDA/FluxReconstructionCUDA.hh"
#include "Framework/MethodCommandProvider.hh"
#include "Framework/VarSetListT.hh"
#include "NavierStokes/Euler2DVarSetT.hh"
#include "NavierStokes/Euler2DConsT.hh"

#include "FluxReconstructionMethod/LaxFriedrichsFlux.hh"
#include <stdio.h>

//////////////////////////////////////////////////////////////////////////////

using namespace COOLFluiD::Framework;
using namespace COOLFluiD::Common;
using namespace COOLFluiD::Config;
using namespace COOLFluiD::Physics::NavierStokes;

//////////////////////////////////////////////////////////////////////////////

namespace COOLFluiD {

    namespace FluxReconstructionMethod {

//////////////////////////////////////////////////////////////////////////////

#define FR_EULER_RHS_PROV(__dim__,__svars__,__uvars__,__nbBThreads__,__providerName__) \
MethodCommandProvider<ConvRHSFluxReconstructionCUDA<LaxFriedrichsFlux, \
                      VarSetListT<Euler##__dim__##__svars__##T, Euler##__dim__##__uvars__##T>, __nbBThreads__>, \
		      FluxReconstructionSolverData,FluxReconstructionCUDAModule>	\
FR_RhsEuler##__dim__##__svars__##__uvars__##__nbBThreads__##Provider(__providerName__);
// 48 block threads (default)
FR_EULER_RHS_PROV(2D, Cons, Cons, 48, "EulerFRLaxFriedrichs2DCons")
//FR_EULER_RHS_PROV(3D, Cons, Cons, 48, "EulerFRLaxFried3DCons")
//FR_NS_RHS_PROV(2D, ProjectionCons, ProjectionPrim, 48, "CellLaxFriedMHD2DPrim")
//FR_NS_RHS_PROV(3D, ProjectionCons, ProjectionPrim, 48, "CellLaxFriedMHD3DPrim")
#undef FR_EULER_RHS_PROV

//////////////////////////////////////////////////////////////////////////////

template <typename PHYS>
HOST_DEVICE inline void setState(CFreal* state, CFreal* statePtr)
{
  // copy the state node data to shared memory
  //for (CFuint i = 0; i < PHYS::DIM; ++i) {node[i] = nodePtr[i];}
  // copy the state data to shared memory
  for (CFuint i = 0; i < PHYS::NBEQS; ++i) {state[i] = statePtr[i];} 
}
      
//////////////////////////////////////////////////////////////////////////////

template <typename PHYS>
HOST_DEVICE void setFluxData(const CFuint stateID, const CFuint cellID, 
			     KernelData<CFreal>* kd, FluxData<PHYS>* fd, const CFuint iSol)
{
  fd->setStateID(LEFT, stateID);
  CFreal* statePtrR = &kd->states[stateID*PHYS::NBEQS];  

  setState<PHYS>(fd->getState(iSol), statePtrR);

  fd->setNbSolPnts(kd->nbSolPnts);
}

//////////////////////////////////////////////////////////////////////////////

template <typename T, CFuint SIZE>
void print(const std::string& name, T* array) 
{
  CFLog(INFO, name << " = \t");
  for (CFuint i = 0; i < SIZE; ++i) {
    CFLog(INFO, array[i] << " ");
  }
  CFLog(INFO, "\n");
}

//////////////////////////////////////////////////////////////////////////////

//template <typename MODEL>
//HOST_DEVICE void computeFaceCentroid(const CellData::Itr* cell, const CFuint faceIdx, 
//				     const CFreal* nodes, CFreal* midFaceCoord)
//{  
//  CudaEnv::CFVecSlice<CFreal, MODEL::DIM> coord(midFaceCoord);
//  coord = 0.;
//  const CFuint nbFaceNodes = cell->getNbFaceNodes(faceIdx);
//  const CFreal ovNbFaceNodes = 1./(static_cast<CFreal>(nbFaceNodes));
//  for (CFuint n = 0; n < nbFaceNodes; ++n) {
//    const CFuint cellNodeID = cell->getNodeID(faceIdx, n);
//    const CFuint nodeID = cell->getNodeID(faceIdx,n);
//    const CFreal* faceNode = &nodes[nodeID*MODEL::DIM];
//    for (CFuint d = 0; d < MODEL::DIM; ++d) {
//      coord[d] += faceNode[d];
//    }
//  }
//  coord *= ovNbFaceNodes;
//}

//////////////////////////////////////////////////////////////////////////////

//template <typename PHYS, typename POLYREC>
//__global__ void computeGradientsKernel(typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
//				       const CFuint nbCells,
//				       CFreal* states, 
//				       CFreal* nodes,
//				       CFreal* centerNodes,
//				       CFreal* ghostStates,
//				       CFreal* ghostNodes,
//				       CFreal* uX,
//				       CFreal* uY,
//				       CFreal* uZ,
//				       CFreal* limiter,
//				       CFreal* updateCoeff, 
//				       CFreal* rhs,
//				       CFreal* normals,
//				       CFint* isOutward,
//				       const CFuint* cellInfo,
//				       const CFuint* cellStencil,
//				       const CFuint* cellFaces,
//				       const CFuint* cellNodes,
//				       const CFint*  neighborTypes,
//				       const Framework::CellConn* cellConn)
//{    
//  // each thread takes care of computing the gradient for one single cell
//  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
//  
//  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
//  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
//  // dcor->init(gdcor);
//  
//  if (cellID < nbCells) {    
//    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
//			   rhs, normals, uX, uY, uZ, isOutward);
//    
//    // compute and store cell gradients at once 
//    POLYREC polyRec(dcor);
//    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
//    CellData::Itr cell = cells.getItr(cellID);
//    polyRec.computeGradients(&states[cellID*PHYS::NBEQS], &centerNodes[cellID*PHYS::DIM], &kd, &cell);
//  }
//}
      
//////////////////////////////////////////////////////////////////////////////

//template <typename PHYS, typename POLYREC, typename LIMITER>
//__global__ void computeLimiterKernel(typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol,
//				     typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
//				     const CFuint nbCells,
//				     CFreal* states, 
//				     CFreal* nodes,
//				     CFreal* centerNodes,
//				     CFreal* ghostStates,
//				     CFreal* ghostNodes,
//				     CFreal* uX,
//				     CFreal* uY,
//				     CFreal* uZ,
//				     CFreal* limiter,
//				     CFreal* updateCoeff, 
//				     CFreal* rhs,
//				     CFreal* normals,
//				     CFint* isOutward,
//				     const CFuint* cellInfo,
//				     const CFuint* cellStencil,
//				     const CFuint* cellFaces,
//				     const CFuint* cellNodes,
//				     const CFint*  neighborTypes,
//				     const Framework::CellConn* cellConn)
//{    
//  // each thread takes care of computing the gradient for one single cell
//  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
//  
//  // __shared__ typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE> s_dcol[32];
//  // typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol = &s_dcol[threadIdx.x];
//  // dcol->init(gdcol);
//  
//  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
//  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
//  // dcor->init(gdcor);
//  
//  if (cellID < nbCells) {    
//    // compute all cell quadrature points at once (size of this array is overestimated)
//    CFreal midFaceCoord[PHYS::DIM*PHYS::DIM*2];
//    
//    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
//    CellData::Itr cell = cells.getItr(cellID);
//    const CFuint nbFacesInCell = cell.getNbFacesInCell();
//    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
//      computeFaceCentroid<PHYS>(&cell, f, nodes, &midFaceCoord[f*PHYS::DIM]);
//    }
//    
//    // compute cell-based limiter at once
//    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
//			   rhs, normals, uX, uY, uZ, isOutward);
//    LIMITER limt(dcol);
//    
//    if (dcor->currRes > dcor->limitRes && (dcor->limitIter > 0 && dcor->currIter < dcor->limitIter)) {	
//      limt.limit(&kd, &cell, &midFaceCoord[0], &limiter[cellID*PHYS::NBEQS]);
//    }
//    else {
//      if (!dcor->freezeLimiter) {
//	// historical modification of the limiter
//	CudaEnv::CFVec<CFreal,PHYS::NBEQS> tmpLimiter;
//	limt.limit(&kd, &cell, &midFaceCoord[0], &tmpLimiter[0]);
//	CFuint currID = cellID*PHYS::NBEQS;
//	for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar, ++currID) {
//	  limiter[currID] = min(tmpLimiter[iVar],limiter[currID]);
//	}
//      }
//    }
//  }
//}
  
//////////////////////////////////////////////////////////////////////////////

template <typename SCHEME, typename PHYS>
__global__ void computeStateLocalRHSKernel(typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof,
                                  typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
                                  const CFuint nbCells,
                                  const CFreal resFactor,
				  CFreal* states, 
                                  CFreal* updateCoeff, 
				  CFreal* rhs,
                                  CFreal* solPntNormals,
                                  CFreal* flxPntNormals,
                                  CFint* faceDir,
                                  const CFuint nbSolPnts,
                                  const CFuint nbrFaces,
                                  const CFuint* faceFlxPntConn,
				  const CFuint* cellInfo,
                                  const CFuint* stateIDs,
                                  const CFuint* neighbCellIDs,
                                  const CFuint* neighbFaceIDs,
                                  const CFuint dim,
                                  const CFuint nbrEqs,
                                  const CFuint nbrFlxPnts,
                                  const CFuint nbrSolSolDep,
                                  const CFuint* solSolDep,
                                  const CFuint nbrSolFlxDep,
                                  const CFuint* solFlxDep,
                                  const CFuint nbrFlxSolDep,
                                  const CFuint* flxSolDep,
                                  const CFreal* solPolyDerivAtSolPnts,
                                  const CFreal* solPolyValsAtFlxPnts,
                                  const CFuint* flxPntFlxDim,
                                  const CFreal* corrFctDiv,
                                  const CFreal* faceIntCoeff)
{    
  // one thread per cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (cellID < nbCells) 
  { 
    // current kernel data
    KernelData<CFreal> kd (nbCells, states, updateCoeff, rhs, solPntNormals, flxPntNormals, faceDir, nbSolPnts);

    // current flux data
    FluxData<typename SCHEME::MODEL> currFd; 

    // initialize flux data
    currFd.initialize();
    
    // physical model
    typename SCHEME::MODEL pmodel(dcop);
    SCHEME fluxScheme(dcof);
    
    // current cell data
    CellData cells(nbCells, cellInfo, stateIDs, neighbCellIDs, neighbFaceIDs, nbrFaces, nbSolPnts);
    
    // get current cell
    CellData::Itr cell = cells.getItr(cellID);
          
    const CFuint nbFlxPntFlx = SCHEME::MODEL::NBEQS*8;
    
    const CFuint nbFaceFlxPntFlx = SCHEME::MODEL::NBEQS*2;
   
    const CFuint nbrFaceFlxPnts = 2;

    const CFuint totNbrFlxPnts = 8;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx> flxPntFlx;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx> flxPntSol;
    
    flxPntFlx = 0.0;
    
    flxPntSol = 0.0;

    for (CFuint iFlx = 0; iFlx < nbrFaceFlxPnts; ++iFlx)
    {
//
      currFd.setFaceIntegrationCoef(iFlx,faceIntCoeff[iFlx]);
//if (cellID == 11) printf("iFlx , coeff\n");
    }
 //     if (cellID == 1) printf("iFlx , coeff\n");
//if (cellID == 11) printf("hello %d\n", 0);
    // loop over sol pnts to compute flux
    for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
    {
//ok
      // get current state ID
      const CFuint stateID = cell.getStateID(iSolPnt);
      //printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
    //if (cellID == 0) printf("GPUstate: %f %f %f %f\n", kd.states[0], kd.states[1], kd.states[2], kd.states[3]);
    
      setFluxData(stateID, cellID, &kd, &currFd, iSolPnt);

      const CFuint nbNormals = PHYS::DIM*PHYS::DIM;

      CudaEnv::CFVecSlice<CFreal,nbNormals> n(&(kd.solPntNormals[stateID*nbNormals]));

      CudaEnv::CFVecSlice<CFreal,nbNormals> nFd(currFd.getScaledNormal(iSolPnt));
      
      for (CFuint i = 0; i < nbNormals; ++i) 
      {
        nFd[i] = n[i];
      }

      // get the flux
      fluxScheme.prepareComputation(&currFd, &pmodel);
//      if (cellID == 11) printf("before LF cell\n");
      fluxScheme(&currFd, &pmodel, false, 1, iSolPnt, cellID);
//      if (cellID == 11) printf("after LF cell\n");
//      // loop over sol pnts to compute flux
//      for (CFuint iDim = 0; iDim < dim; ++iDim)
//      {
//        if (cellID == 0) printf("HERE4 iSol: %d, iDim: %d, flux: %f %f %f %f \n", iSolPnt, iDim, currFd.getFlux(iSolPnt, iDim)[0], currFd.getFlux(iSolPnt, iDim)[1], currFd.getFlux(iSolPnt, iDim)[2], currFd.getFlux(iSolPnt, iDim)[3]);
//      }
    }

    for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
    {
      // get current state ID
      const CFuint stateID = cell.getStateID(iSolPnt);

      setFluxData(stateID, cellID, &kd, &currFd, iSolPnt);

      // Loop over solution pnts to count the factor of all sol pnt polys
      for (CFuint jSolPnt = 0; jSolPnt < nbrSolSolDep; ++jSolPnt)
      { 
        const CFuint jSolIdx = solSolDep[iSolPnt*nbrSolSolDep+jSolPnt]; //(*m_solSolDep)[iSolPnt][jSolPnt];

        // get current vector slice out of rhs
        CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> res(&rhs[stateID*SCHEME::MODEL::NBEQS]);
//if (cellID == 11) printf("resID: %d\n", stateID);
        // Loop over deriv directions and sum them to compute divergence
        for (CFuint iDir = 0; iDir < dim; ++iDir)
        {
          const CFreal polyCoef = solPolyDerivAtSolPnts[iSolPnt*dim*nbSolPnts+iDir*nbSolPnts+jSolIdx];//(*m_solPolyDerivAtSolPnts)[jSolPnt][iDir][iSolIdx]; 

          //if (cellID == 0) printf("polyCoef: %f\n", polyCoef);
          
          // Loop over conservative fluxes 
          for (CFuint iEq = 0; iEq < nbrEqs; ++iEq)
          {
//if (cellID == 11 && abs(polyCoef*(currFd.getFlux(jSolIdx, iDir)[iEq])) > 1e-8) printf("State: %d, jSol: %d, iDir: %d, var: %d, res before: %f\n",iSolPnt,jSolIdx,iDir,iEq,res[iEq]);

            // Store divFD in the vector that will be divFC
            res[iEq] -= polyCoef*(currFd.getFlux(jSolIdx, iDir)[iEq])*resFactor;

//if (cellID == 11 && abs(polyCoef*(currFd.getFlux(jSolIdx, iDir)[iEq])) > 1e-8) printf("State: %d, jSol: %d, iDir: %d, var: %d, up: %f, poly: %f, flx: %f\n",iSolPnt,jSolIdx,iDir,iEq,polyCoef*(currFd.getFlux(iSolPnt, iDir)[iEq]),polyCoef,currFd.getFlux(jSolIdx, iDir)[iEq]);  
            //if (cellID == 0) printf("res %f \n", res[iEq]);
	  }
        }
      }
    }

    // extrapolate the fluxes to the flux points
    for (CFuint iFlxPnt = 0; iFlxPnt < nbrFlxPnts; ++iFlxPnt)
    {
      const CFuint dim = flxPntFlxDim[iFlxPnt];

      // loop over sol pnts to compute flux
      for (CFuint iSolPnt = 0; iSolPnt < nbrFlxSolDep; ++iSolPnt)
      {
        const CFuint solIdx = flxSolDep[iFlxPnt*nbrFlxSolDep + iSolPnt];

        // Loop over conservative fluxes 
        for (CFuint iEq = 0; iEq < nbrEqs; ++iEq)
        {
          flxPntFlx[iFlxPnt*nbrEqs+iEq] += solPolyValsAtFlxPnts[iFlxPnt*nbSolPnts+solIdx]*currFd.getFlux(solIdx, dim)[iEq];

          flxPntSol[iFlxPnt*nbrEqs+iEq] += solPolyValsAtFlxPnts[iFlxPnt*nbSolPnts+solIdx]*kd.states[iEq];          
        }
      }
    }

    // set extrapolated states
    for (CFuint iState = 0; iState < nbrFlxPnts; ++iState)
    {
      for (CFuint iEq = 0; iEq < PHYS::NBEQS; ++iEq) 
      {
        currFd.getLstate(iState)[iEq] = flxPntSol[iState*PHYS::NBEQS+iEq];
        //printf("stateL %d: %f\n", iEq, currFd.getLstate(iState)[iEq]);
      } 
    }

    for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
    {
      // get current state ID
      const CFuint stateID = cell.getStateID(iSolPnt);

      // get current vector slice out of rhs
      CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> res(&rhs[stateID*SCHEME::MODEL::NBEQS]);

      // add divhFD to the residual updates
      for (CFuint iFlxPnt = 0; iFlxPnt < nbrSolFlxDep; ++iFlxPnt)
      {
        const CFuint flxIdx = solFlxDep[iSolPnt*nbrSolFlxDep+iFlxPnt];

        // get the divergence of the correction function
        const CFreal divh = corrFctDiv[iSolPnt*nbrFlxPnts+flxIdx];

        // Fill in the corrections
        for (CFuint iVar = 0; iVar < nbrEqs; ++iVar)
        {
          res[iVar] += flxPntFlx[flxIdx*nbrEqs+iVar] * divh * resFactor;
//if (cellID==11 && abs(flxPntFlx[flxIdx*nbrEqs+iVar] * divh) > 1e-8) printf("State: %d, flx: %d, var: %d, update: %f\n",iSolPnt,flxIdx,iVar,flxPntFlx[flxIdx*nbrEqs+iVar] * divh);  
//if (cellID==11 && abs(flxPntFlx[flxIdx*nbrEqs+iVar] * divh) > 1e-8) printf("res after: %f\n",res[iVar]);  
        }
      }
    }

    // reset flx pnt fluxes  
    flxPntFlx = 0.0;
    
    //const CFuint nbFaceFlxPnts = nbFlxPntFlx/nbrFaces;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx> flxPntSolNeighb;
    
    for (CFuint iFlxPnt = 0; iFlxPnt < nbFlxPntFlx; ++iFlxPnt) {flxPntSolNeighb[iFlxPnt] = 0.0;}

    // current neighb cell data
    CellData cells2(nbCells, cellInfo, stateIDs, neighbCellIDs, neighbFaceIDs, nbrFaces, nbSolPnts);

    for (CFuint iFace = 0; iFace < nbrFaces; ++iFace)
    {

      const CFuint neighbCellID = cell.getNeighbCellID(iFace);  

      // get current cell
      CellData::Itr cell2 = cells2.getItr(neighbCellID);

      CFuint jFaceIdx = 100;

      for (CFuint jFace = 0; jFace < nbrFaces; ++jFace)
      {
        if (cell2.getNeighbCellID(jFace) == cellID)
        {
          jFaceIdx = jFace; 
          break;
        }
      }
//if (cellID == 11) printf("bbbb\n");
      if (jFaceIdx != 100)
      {
        const CFuint faceID = cell.getNeighbFaceID(iFace);

      // loop over face flx pnts
      for (CFuint iFlxPnt = 0; iFlxPnt < nbrFaceFlxPnts; ++iFlxPnt)
      { 
        const CFuint flxIdx = faceFlxPntConn[iFace*nbrFaceFlxPnts+iFlxPnt];
        const CFuint jFlxIdx = faceFlxPntConn[jFaceIdx*nbrFaceFlxPnts+iFlxPnt];

        // loop over sol pnts to compute sol at flx pnt
        for (CFuint iSolPnt = 0; iSolPnt < nbrFlxSolDep; ++iSolPnt)
        {

          const CFuint solIdx = flxSolDep[jFlxIdx*nbrFlxSolDep+iSolPnt]; 
           //printf("flxIdx: %d\n", flxIdx);

          // Loop over conservative vars 
          for (CFuint iEq = 0; iEq < nbrEqs; ++iEq)
          {
            //printf("state %d before: %d, solID: %d, state: %f\n", iEq, cell2.getStateID(solIdx), flxIdx*nbrEqs+iEq, 0.0);
            flxPntSolNeighb[flxIdx*nbrEqs+iEq] += solPolyValsAtFlxPnts[jFlxIdx*nbSolPnts+solIdx]*states[cell2.getStateID(solIdx)*SCHEME::MODEL::NBEQS+iEq];
          }
        }

        // get current state ID
        //const CFuint neighbStateID = cell.getNeighbStateID(iFace,iSolPnt);
        
        //const CFuint neighbCellID = cell.getNeighbCellID(iFace);

        for (CFuint iEq = 0; iEq < PHYS::NBEQS; ++iEq) 
        {
          currFd.getRstate(flxIdx)[iEq] = flxPntSolNeighb[flxIdx*PHYS::NBEQS+iEq];
          //printf("stateR %d: %f\n", iEq, currFd.getRstate(flxIdx)[iEq]);
        } 

        CudaEnv::CFVecSlice<CFreal,PHYS::DIM> n(&(kd.flxPntNormals[faceID*nbrFaceFlxPnts*PHYS::DIM+iFlxPnt*PHYS::DIM]));

        CudaEnv::CFVecSlice<CFreal,PHYS::DIM> nFd(currFd.getFlxScaledNormal(flxIdx));
//if (cellID == 11) printf("faceID: %d, flx idx: %d, normal: %f, %f\n",faceID,flxIdx,n[0],n[1]);
        for (CFuint i = 0; i < PHYS::DIM; ++i) 
        {
          nFd[i] = n[i];
        }

//      if (cellID == 11) printf("aaa\n");
        // get the flux
        fluxScheme.prepareComputation(&currFd, &pmodel);
        //printf("flxIdx: %d\n", flxIdx);
//if (cellID == 11) printf("before LF face\n");

        fluxScheme(&currFd, &pmodel, true, iFlxPnt, flxIdx, cellID);
//if (cellID == 11) printf("after LF face\n");

        // extrapolate the fluxes to the flux points
        for (CFuint iSolPnt = 0; iSolPnt < nbrFlxSolDep; ++iSolPnt)
        {     
          const CFuint solIdx = flxSolDep[flxIdx*nbrFlxSolDep+iSolPnt];

          // get current state ID
          const CFuint stateID = cell.getStateID(solIdx);

          // get current vector slice out of rhs
          CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> res(&rhs[stateID*SCHEME::MODEL::NBEQS]);   
          
          // divergence of the correction function
          const CFreal divh = corrFctDiv[solIdx*nbrFlxPnts+flxIdx] * faceDir[cellID*totNbrFlxPnts+flxIdx];
//if (cellID == 11) printf("State %d, flx %d, divh %f\n",solIdx,flxIdx,divh);
          // Fill in the corrections
          for (CFuint iVar = 0; iVar < nbrEqs; ++iVar)
          {
//if (cellID==1752) printf("resBefore: %f\n",res[iVar]);
            res[iVar] -= currFd.getInterfaceFlux(flxIdx)[iVar] * divh * resFactor;
//if (cellID == 11) printf("var %d, fI %f\n",iVar,currFd.getInterfaceFlux(flxIdx)[iVar]);
//if (cellID==1752) printf("resID: %d, State: %d, flx: %d, var: %d, updateFace: %f, res: %f\n",stateID*SCHEME::MODEL::NBEQS+iVar,solIdx,flxIdx,iVar,-currFd.getInterfaceFlux(flxIdx)[iVar] * divh, res[iVar]); 
          }
        }
      }

      CFreal* waveSpeedUpd = currFd.getUpdateCoeff();

      for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
      {  
        // get current state ID
        const CFuint stateID = cell.getStateID(iSolPnt);

        updateCoeff[stateID] += *waveSpeedUpd;

//if (cellID == 11) printf("iSol: %d, upd %f\n",iSolPnt,updateCoeff[stateID]);
      }
 
      currFd.resetUpdateCoeff();

//if (cellID == 11) printf("end face \n",cellID);
      }
    }
    //if (cellID == 0) printf("end sol pnt \n",cellID);
  }
}
  
//////////////////////////////////////////////////////////////////////////////

//template <typename SCHEME, typename POLYREC, typename LIMITER>
//void computeFluxCPU(CFuint nbThreadsOMP,
//		    typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof,
//		    typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
//		    typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol,
//		    typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
//		    const CFuint nbCells,
//		    CFreal* states, 
//		    CFreal* nodes,
//		    CFreal* centerNodes,
//		    CFreal* ghostStates,
//		    CFreal* ghostNodes,
//		    CFreal* uX,
//		    CFreal* uY,
//		    CFreal* uZ,
//		    CFreal* limiter,
//		    CFreal* updateCoeff, 
//		    CFreal* rhs,
//		    CFreal* normals,
//		    CFint* isOutward,
//		    const CFuint* cellInfo,
//		    const CFuint* cellStencil,
//		    const CFuint* cellFaces,
//		    const CFuint* cellNodes,
//		    const CFint* neighborTypes,
//		    const Framework::CellConn* cellConn)
//{ 
//  typedef typename SCHEME::MODEL PHYS;
//  
//  FluxData<PHYS> fd;
//#ifndef CF_HAVE_OMP  
//  fd.initialize();
//  FluxData<PHYS>* currFd = &fd;
//  cf_assert(currFd != CFNULL);
//#endif
//  POLYREC polyRec(dcor);
//  SCHEME fluxScheme(dcof);
//  LIMITER limt(dcol);
//  PHYS pmodel(dcop);
//  
//  CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
//  KernelData<CFreal> kd(nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
//			rhs, normals, uX, uY, uZ, isOutward);
//  
//  CFreal midFaceCoord[PHYS::DIM*PHYS::DIM*2];
//  CudaEnv::CFVec<CFreal,PHYS::NBEQS> tmpLimiter;
//
//#ifdef CF_HAVE_OMP
//  //const CFuint nThr = omp_get_num_procs();
//  // omp_set_num_threads(nbThreadsOMP);
//#pragma omp num_thread(nbThreadsOMP) parallel private(polyRec) private(fd)
//{
//  #pragma omp for
//#endif 
//  // compute the cell-based gradients
//  for (CFuint cellID = 0; cellID < nbCells; ++cellID) {
//#ifdef CF_HAVE_OMP
//    fd.initialize();
//    FluxData<PHYS>* currFd = &fd;
//    cf_assert(currFd != CFNULL);
//#endif 
//    CellData::Itr cell = cells.getItr(cellID);
//    polyRec.computeGradients(&states[cellID*PHYS::NBEQS], &centerNodes[cellID*PHYS::DIM], &kd, &cell);
//  }
//#ifdef CF_HAVE_OMP
//}
//#endif
//
//#ifdef CF_HAVE_OMP  
//#pragma omp num_thread(nbThreadsOMP) parallel private(limt) private(kd)
//{
//  #pragma omp for
//#endif 
//  // compute the cell based limiter 
//  for (CFuint cellID = 0; cellID < nbCells; ++cellID) {
//  // for (CellData::Itr cell = cells.begin(); cell <= cells.end(); ++cell) {
//    CellData::Itr cell = cells.getItr(cellID);
//    // compute all cell quadrature points at once (size of this array is overestimated)
//    const CFuint nbFacesInCell = cell.getNbFacesInCell();
//    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
//      computeFaceCentroid<PHYS>(&cell, f, nodes, &midFaceCoord[f*PHYS::DIM]);
//    }
//    
//    //   const CFuint cellID = cell.getCellID();
//    if (dcor->currRes > dcor->limitRes && (dcor->limitIter > 0 && dcor->currIter < dcor->limitIter)) {	
//      // compute cell-based limiter
//      limt.limit(&kd, &cell, &midFaceCoord[0], &limiter[cellID*PHYS::NBEQS]);
//    }
//    else {
//      if (!dcor->freezeLimiter) {
//	// historical modification of the limiter
//	limt.limit(&kd, &cell, &midFaceCoord[0], &tmpLimiter[0]);
//	CFuint currID = cellID*PHYS::NBEQS;
//	for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar, ++currID) {
//	  limiter[currID] = min(tmpLimiter[iVar],limiter[currID]);
//	}
//      }
//    }
//  }
//#ifdef CF_HAVE_OMP
//}
//
//#pragma omp num_thread(nbThreadsOMP) parallel private(fd) private(kd) private(fluxScheme) private(pmodel)
//{
//  #pragma omp for
//#endif 
//  // compute the fluxes
//  for (CFuint cellID = 0; cellID < nbCells; ++cellID) {
//  //  for (CellData::Itr cell = cells.begin(); cell <= cells.end(); ++cell) {
//#ifdef CF_HAVE_OMP
//    fd.initialize();
//    FluxData<PHYS>* currFd = &fd;
//    cf_assert(currFd != CFNULL);
//#endif
//    // reset the rhs and update coefficients to 0
//   // const CFuint cellID = cell.getCellID();
//    CudaEnv::CFVecSlice<CFreal,PHYS::NBEQS> res(&rhs[cellID*PHYS::NBEQS]);
//    res = 0.;
//    updateCoeff[cellID] = 0.;
//
//    CellData::Itr cell = cells.getItr(cellID);   
//    const CFuint nbFacesInCell = cell.getNbActiveFacesInCell();
//    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
//      const CFint stype = cell.getNeighborType(f);
//      
//      if (stype != 0) { // skip all partition faces
//	const CFuint stateID =  cell.getNeighborID(f);
//	setFluxData(f, stype, stateID, cellID, &kd, currFd, cellFaces);
//	
//	// compute face quadrature points (centroid)
//	CFreal* faceCenters = &midFaceCoord[f*PHYS::DIM];
//	computeFaceCentroid<PHYS>(&cell, f, nodes, faceCenters);
//	
//	// extrapolate solution on quadrature points on both sides of the face
//	polyRec.extrapolateOnFace(currFd, faceCenters, uX, uY, uZ, limiter);
//        fluxScheme.prepareComputation(currFd, &pmodel);
//	fluxScheme(currFd, &pmodel); // compute the convective flux across the face
//	
//	for (CFuint iEq = 0; iEq < PHYS::NBEQS; ++iEq) {
//	  const CFreal value = currFd->getResidual()[iEq];
//	  res[iEq] -= value;  // update the residual 
//	}
//	
//	// update the update coefficient
//	updateCoeff[cellID] += currFd->getUpdateCoeff();
//      }
//    }
//  }
//#ifdef CF_HAVE_OMP
//} 
//#endif
//}

//////////////////////////////////////////////////////////////////////////////

template <typename SCHEME, typename PHYSICS, CFuint NB_BLOCK_THREADS>
void ConvRHSFluxReconstructionCUDA<SCHEME,PHYSICS,NB_BLOCK_THREADS>::execute()
{
  using namespace COOLFluiD::Framework;
  using namespace COOLFluiD::Common;
  
  CFTRACEBEGIN;
  
  CFLog(VERBOSE, "ConvRHSFluxReconstructionCUDA::execute() START\n");
  
  // get the elementTypeData
  SafePtr< vector<ElementTypeData> > elemType = MeshDataStack::getActive()->getElementTypeData();

  // get InnerCells TopologicalRegionSet
  SafePtr<TopologicalRegionSet> cells = MeshDataStack::getActive()->getTrs("InnerCells");

  // get the geodata of the geometric entity builder and set the TRS
  StdTrsGeoBuilder::GeoData& geoDataCell = m_cellBuilder->getDataGE();
  geoDataCell.trs = cells;
  
  // get InnerFaces TopologicalRegionSet
  SafePtr<TopologicalRegionSet> faces = MeshDataStack::getActive()->getTrs("InnerFaces");

  // get the face start indexes
  vector< CFuint >& innerFacesStartIdxs = getMethodData().getInnerFacesStartIdxs();

  // get number of face orientations
  const CFuint nbrFaceOrients = innerFacesStartIdxs.size()-1;

  // get the geodata of the face builder and set the TRSs
  FaceToCellGEBuilder::GeoData& geoDataFace = m_faceBuilder->getDataGE();
  geoDataFace.cellsTRS = cells;
  geoDataFace.facesTRS = faces;
  geoDataFace.isBoundary = false;
  
  // loop over element types, for the moment there should only be one
  const CFuint nbrElemTypes = elemType->size();
  cf_assert(nbrElemTypes == 1);
  
  // get start and end indexes for this type of element
  cf_assert((*elemType)[0].getStartIdx() == 0);
  const CFuint nbCells   = (*elemType)[0].getEndIdx();
  cf_assert(nbCells > 0);
  
  //initializeComputationRHS();

  const CFuint nbStates = socket_states.getDataHandle().size();
  cf_assert(nbStates > 0);

  CFLog(INFO, "nbCells: " << nbCells << ", nbStates: " << nbStates << "\n");

  DataHandle<CFreal> updateCoeff = socket_updateCoeff.getDataHandle();
  DataHandle<CFreal> rhs = socket_rhs.getDataHandle(); 
  DataHandle<CFreal> solPntNormals = socket_solPntNormals.getDataHandle(); 
  DataHandle<CFreal> flxPntNormals = socket_flxPntNormals.getDataHandle(); 
  DataHandle<CFint> faceDir = socket_faceDir.getDataHandle();  

  SafePtr<SCHEME> lf  = getMethodData().getRiemannFlux().d_castTo<SCHEME>();
  SafePtr<typename PHYSICS::PTERM> phys = PhysicalModelStack::getActive()->getImplementor()->
    getConvectiveTerm().d_castTo<typename PHYSICS::PTERM>();
  
#ifdef CF_HAVE_CUDA
  typedef typename SCHEME::template DeviceFunc<GPU, PHYSICS> FluxScheme;  
#else
  typedef typename SCHEME::template DeviceFunc<CPU, PHYSICS> FluxScheme;
#endif 
  
  if (m_onGPU) 
  {
#ifdef CF_HAVE_CUDA

    CudaEnv::CudaTimer& timer = CudaEnv::CudaTimer::getInstance();
    timer.start();
    
    // copy of data that change at every iteration
    socket_states.getDataHandle().getGlobalArray()->put(); 
    socket_rhs.getDataHandle().getLocalArray()->put(); 
    socket_updateCoeff.getDataHandle().getLocalArray()->put();
    socket_faceDir.getDataHandle().getLocalArray()->put();
    
    CFLog(VERBOSE, "nb normals: " << socket_solPntNormals.getDataHandle().size() << ", n0: " << socket_solPntNormals.getDataHandle()[0] << "\n");
 
    socket_solPntNormals.getDataHandle().getLocalArray()->put();
    socket_flxPntNormals.getDataHandle().getLocalArray()->put();
    DataHandle<Framework::State*, Framework::GLOBAL > statesI = socket_states.getDataHandle();
     
    CFLog(VERBOSE, "ConvRHSFluxReconstructionCUDA::execute() => CPU-->GPU data transfer took " << timer.elapsed() << " s\n");
    timer.start();
    
    ConfigOptionPtr<SCHEME,  NOTYPE, GPU> dcof(lf);
    ConfigOptionPtr<typename PHYSICS::PTERM, NOTYPE, GPU> dcop(phys);

    const CFuint blocksPerGrid = CudaEnv::CudaDeviceManager::getInstance().getBlocksPerGrid(nbCells);
    const CFuint nThreads = CudaEnv::CudaDeviceManager::getInstance().getNThreads();
    CFLog(VERBOSE, "blocksPerGrid: " << blocksPerGrid << ", threads: " << nThreads << "\n");

//CFuint megabytesToUse = 24;
//size_t newHeapSize = 1024 * 1000 * megabytesToUse;
//hipDeviceSetLimit(hipLimitMallocHeapSize, newHeapSize);
//printf("Adjusted heap size to be %d\n",(int) newHeapSize);

    //dim3 blocks(m_nbBlocksPerGridX, m_nbBlocksPerGridY);
    
    //hipFuncSetCacheConfig(reinterpret_cast<const void*>("computeGradientsKernel"), hipFuncCachePreferL1);

    // get residual factor
    const CFreal resFactor = getMethodData().getResFactor();
    
    // hipFuncSetCacheConfig(reinterpret_cast<const void*>("computeFluxKernel"), hipFuncCachePreferL1);
    CFLog(INFO, "Before Kernel: " << m_faceIntegrationCoefs2[0] << ", " << m_faceIntegrationCoefs2[1] << "\n");
    // compute the convective flux in each cell
    computeStateLocalRHSKernel<FluxScheme,PHYSICS> <<<blocksPerGrid,nThreads>>> 
      (dcof.getPtr(),
       dcop.getPtr(),
       nbCells,
       resFactor,
       socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
       updateCoeff.getLocalArray()->ptrDev(), 
       rhs.getLocalArray()->ptrDev(),
       solPntNormals.getLocalArray()->ptrDev(),
       flxPntNormals.getLocalArray()->ptrDev(),
       faceDir.getLocalArray()->ptrDev(),
       m_nbrSolPnts,
       4,
       m_faceFlxPntConn2.ptrDev(),
       m_cellInfo.ptrDev(),
       m_stateIDs.ptrDev(),
       m_neighbCellIDs.ptrDev(),
       m_neighbFaceIDs.ptrDev(),
       m_dim,
       m_nbrEqs,
       m_nbrFlxPnts,
       m_nbrSolSolDep,
       m_solSolDep2.ptrDev(),
       m_nbrFlxDep,
       m_solFlxDep2.ptrDev(),
       m_nbrSolDep,
       m_flxSolDep2.ptrDev(),
       m_solPolyDerivAtSolPnts2.ptrDev(),
       m_solPolyValsAtFlxPnts2.ptrDev(),
       m_flxPntFlxDim2.ptrDev(),
       m_corrFctDiv2.ptrDev(),
       m_faceIntegrationCoefs2.ptrDev());
    
    hipDeviceSynchronize();
    
    //for (CFuint i = 0; i < m_solPolyValsAtFlxPnts2.size(); ++i) {CFLog(INFO, "thing: " << m_solPolyValsAtFlxPnts2[i] << "\n");}
    
    //CFLog(INFO, "After Kernel, size: " << socket_states.getDataHandle().size() << "\n");
    
    CFLog(VERBOSE, "ConvRHSFluxReconstructionCUDA::execute() => computeFluxKernel took " << timer.elapsed() << " s\n");
    
    //for (CFuint i = 0; i < rhs.size(); ++i) {CFLog(INFO, "res before: " << rhs[i] << "\n");}
    
    //RealVector rhsB;
    //rhsB.resize(rhs.size());
    //for (CFuint i = 0; i < rhs.size(); ++i) {rhsB[i] = rhs[i];}
    
    timer.start();
    rhs.getLocalArray()->get();
    updateCoeff.getLocalArray()->get();
    
    //for (CFuint i = 0; i < rhs.size(); ++i) {CFLog(INFO, "res after: " << rhs[i]-rhsB[i] << "\n");}
    CFLog(VERBOSE, "ConvRHSFluxReconstructionCUDA::execute() => GPU-->CPU data transfer took " << timer.elapsed() << " s\n");
    //CFLog(INFO, "resSize: " << rhs.size() << "\n");
    for (CFuint i = 0; i < rhs.size(); ++i)
    {
      //if (abs(rhs[i]) > 1.0e-10) CFLog(INFO, "res " << i << ": " << rhs[i] << "\n");
    }

  #endif
  }
  else 
  {
  // boolean telling whether there is a diffusive term
  const bool hasDiffTerm = getMethodData().hasDiffTerm() || getMethodData().hasArtificialViscosity();

  // loop over element types, for the moment there should only be one
  for (m_iElemType = 0; m_iElemType < nbrElemTypes; ++m_iElemType)
  {
    // get start and end indexes for this type of element
    const CFuint startIdx = (*elemType)[m_iElemType].getStartIdx();
    const CFuint endIdx   = (*elemType)[m_iElemType].getEndIdx();

    // loop over cells
    for (CFuint elemIdx = startIdx; elemIdx < endIdx; ++elemIdx)
    {
      // build the GeometricEntity
      geoDataCell.idx = elemIdx;
      m_cell = m_cellBuilder->buildGE();

      // get the states in this cell
      m_cellStates = m_cell->getStates();
      
      // if the states in the cell are parallel updatable or the gradients need to be computed, set the cell data
      if ((*m_cellStates)[0]->isParUpdatable() || hasDiffTerm)
      {
	// set the cell data
	setCellData();
      }
      
      // if the states in the cell are parallel updatable, compute the divergence of the discontinuous flx (-divFD+divhFD)
      if ((*m_cellStates)[0]->isParUpdatable())
      {
	// compute the divergence of the discontinuous flux (-divFD+divhFD)
	computeDivDiscontFlx(m_divContFlx);
      
	// update RHS
        updateRHS();
      } 
      
      // if there is a diffusive term, compute the gradients
      if (hasDiffTerm)
      {
	computeGradients();
      }
      
      // print out the residual updates for debugging
      if(m_cell->getID() == 35) //true) //
      {
	CFLog(VERBOSE, "ID  = " << (*m_cellStates)[0]->getLocalID() << "\n");
        CFLog(VERBOSE, "coords  = " << (*m_cellStates)[0]->getCoordinates() << "\n");
        CFLog(VERBOSE, "UpdateTotal = \n");
        // get the datahandle of the rhs
        DataHandle< CFreal > rhs = socket_rhs.getDataHandle();
        for (CFuint iState = 0; iState < m_nbrSolPnts; ++iState)
        {
          CFuint resID = m_nbrEqs*( (*m_cellStates)[iState]->getLocalID() );
          for (CFuint iVar = 0; iVar < m_nbrEqs; ++iVar)
          {
            CFLog(VERBOSE, "" << rhs[resID+iVar] << " ");
          }
          CFLog(VERBOSE,"\n");
          DataHandle<CFreal> updateCoeff = socket_updateCoeff.getDataHandle();
          CFLog(VERBOSE, "UpdateCoeff: " << updateCoeff[(*m_cellStates)[iState]->getLocalID()] << "\n");
	  CFLog(VERBOSE, "state " << iState << ": " << *(((*m_cellStates)[iState])->getData()) << "\n");
        }
      }
      
      if(m_cell->getID() == 35 && hasDiffTerm)
      {
	// get the gradients
        DataHandle< vector< RealVector > > gradients = socket_gradients.getDataHandle();

        for (CFuint iState = 0; iState < m_nbrSolPnts; ++iState)
        {
	  CFuint solID = ((*m_cellStates)[iState])->getLocalID();
          for (CFuint iGrad = 0; iGrad < m_nbrEqs; ++iGrad)
          {
	    CFLog(VERBOSE, "total gradient " << iGrad << " of  " << iState << ": " << gradients[solID][iGrad] << "\n");
          } 
        }
        for (CFuint iState = 0; iState < m_nbrSolPnts; ++iState)
        {
	  CFLog(VERBOSE, "state " << iState << ": " << *(((*m_cellStates)[iState])->getData()) << "\n");
	}
      }
      
      //release the GeometricEntity
      m_cellBuilder->releaseGE();
    }
  }
  
  //// Loop over faces to calculate fluxes and interface fluxes in the flux points
  
  // loop over different orientations
  for (m_orient = 0; m_orient < nbrFaceOrients; ++m_orient)
  {
    CFLog(VERBOSE, "Orient = " << m_orient << "\n");
    // start and stop index of the faces with this orientation
    const CFuint faceStartIdx = innerFacesStartIdxs[m_orient  ];
    const CFuint faceStopIdx  = innerFacesStartIdxs[m_orient+1];

    // loop over faces with this orientation
    for (CFuint faceID = faceStartIdx; faceID < faceStopIdx; ++faceID)
    {
      // build the face GeometricEntity
      geoDataFace.idx = faceID;
      m_face = m_faceBuilder->buildGE();

      // get the neighbouring cells
      m_cells[LEFT ] = m_face->getNeighborGeo(LEFT );
      m_cells[RIGHT] = m_face->getNeighborGeo(RIGHT);

      // get the states in the neighbouring cells
      m_states[LEFT ] = m_cells[LEFT ]->getStates();
      m_states[RIGHT] = m_cells[RIGHT]->getStates();

      // if one of the neighbouring cells is parallel updatable or if the gradients have to be computed, set the bnd face data
      if ((*m_states[LEFT ])[0]->isParUpdatable() || (*m_states[RIGHT])[0]->isParUpdatable() || hasDiffTerm)
      {
	// set the bnd face data
        setFaceData(m_face->getID());//faceID
if (m_cells[LEFT ]->getID()==11 || m_cells[RIGHT ]->getID()==11) CFLog(INFO, "faceID: " << m_face->getID() << "\n");
	// compute the states in the flx pnts
        computeFlxPntStates();

	// compute the interface flux
	computeInterfaceFlxCorrection();
          
	// compute the wave speed updates
        computeWaveSpeedUpdates(m_waveSpeedUpd);

        // update the wave speed
        updateWaveSpeed();
      }
	
	// if one of the neighbouring cells is parallel updatable, compute the correction flux
      if ((*m_states[LEFT ])[0]->isParUpdatable() || (*m_states[RIGHT])[0]->isParUpdatable())
      {
	
	// compute the correction for the left neighbour
	computeCorrection(LEFT, m_divContFlxL);
	
	// compute the correction for the right neighbour
	computeCorrection(RIGHT, m_divContFlxR);
	
	// update RHS
	updateRHSBothSides();
      }
      
      // if there is a diffusive term, compute the gradients
      if (hasDiffTerm)
      {
	// compute the face correction term of the corrected gradients
        computeGradientFaceCorrections();
      }

      // release the GeometricEntity
      m_faceBuilder->releaseGE();
    }
  }

DataHandle< CFreal > rhs = socket_rhs.getDataHandle();
//CFLog(INFO, "resSize: " << rhs.size() << "\n");
for (CFuint i = 0; i < rhs.size(); ++i)
 {
    //if (abs(rhs[i]) > 1.0e-10) CFLog(INFO, "res " << i << ": " << rhs[i] << "\n");
  }
  }
  
  //finalizeComputationRHS();
  
  CFLog(VERBOSE, "ConvRHSFluxReconstructionCUDA::execute() END\n");
  
  CFTRACEEND;
}

//////////////////////////////////////////////////////////////////////////////

    } // namespace FluxReconstructionMethod

} // namespace COOLFluiD
