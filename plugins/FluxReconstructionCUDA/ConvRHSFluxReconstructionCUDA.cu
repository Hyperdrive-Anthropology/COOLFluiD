#include "hip/hip_runtime.h"
#include "FluxReconstructionCUDA/ConvRHSFluxReconstructionCUDA.hh"
#include "Framework/MeshData.hh"
#include "Framework/CellConn.hh"
#include "Config/ConfigOptionPtr.hh"
#include "Framework/CudaDeviceManager.hh"
#include "Common/CUDA/CFVec.hh"
#include "Framework/CudaTimer.hh"

//#include "FiniteVolume/FluxData.hh"
#include "FluxReconstructionMethod/KernelData.hh"
#include "FluxReconstructionMethod/CellData.hh"

#include "FluxReconstructionCUDA/FluxReconstructionCUDA.hh"
#include "Framework/MethodCommandProvider.hh"
#include "Framework/VarSetListT.hh"
#include "NavierStokes/EulerVarSet.hh"
#include "NavierStokes/Euler2DCons.hh"
#include "NavierStokes/Euler3DCons.hh"

#include "FluxReconstructionMethod/LaxFriedrichsFlux.hh"


//////////////////////////////////////////////////////////////////////////////

using namespace COOLFluiD::Framework;
using namespace COOLFluiD::Common;
using namespace COOLFluiD::Config;
using namespace COOLFluiD::Physics::NavierStokes;

//////////////////////////////////////////////////////////////////////////////

namespace COOLFluiD {

    namespace FluxReconstructionMethod {

//////////////////////////////////////////////////////////////////////////////

#define FR_EULER_RHS_PROV(__dim__,__svars__,__uvars__,__nbBThreads__,__providerName__) \
MethodCommandProvider<ConvRHSFluxReconstructionCUDA<LaxFriedrichsFlux, \
                      VarSetListT<Euler##__dim__##__svars__, Euler##__dim__##__uvars__>, __nbBThreads__>, \
		      FluxReconstructionSolverData,FluxReconstructionCUDAModule>	\
FR_RhsEuler##__dim__##__svars__##__uvars__##__nbBThreads__##Provider(__providerName__);
// 48 block threads (default)
FR_EULER_RHS_PROV(2D, Cons, Cons, 48, "EulerFRLaxFried2DCons")
FR_EULER_RHS_PROV(3D, Cons, Cons, 48, "EulerFRLaxFried3DCons")
//FR_NS_RHS_PROV(2D, ProjectionCons, ProjectionPrim, 48, "CellLaxFriedMHD2DPrim")
//FR_NS_RHS_PROV(3D, ProjectionCons, ProjectionPrim, 48, "CellLaxFriedMHD3DPrim")
#undef FR_EULER_RHS_PROV

//////////////////////////////////////////////////////////////////////////////

template <typename PHYS>
HOST_DEVICE inline void setState(CFreal* state, CFreal* statePtr, 
				 CFreal* node, CFreal* nodePtr)
{
  // copy the state node data to shared memory
  for (CFuint i = 0; i < PHYS::DIM; ++i) {node[i] = nodePtr[i];}
  // copy the state data to shared memory
  for (CFuint i = 0; i < PHYS::NBEQS; ++i) {state[i] = statePtr[i];} 
}
      
//////////////////////////////////////////////////////////////////////////////
      
//template <typename PHYS>
//HOST_DEVICE inline void setFaceNormal(FluxData<PHYS>* fd, CFreal* normal)
//{
//  CudaEnv::CFVecSlice<CFreal,PHYS::DIM> n(normal);
//  const CFreal area = n.norm2();
//  fd->setFaceArea(area);
//  const CFreal ovArea = 1./area;
//  CudaEnv::CFVecSlice<CFreal,PHYS::DIM> un(fd->getUnitNormal());
//  for (CFuint i = 0; i < PHYS::DIM; ++i) {
//    un[i] = n[i]*ovArea;
//  }
//}
      
//////////////////////////////////////////////////////////////////////////////

//template <typename PHYS, typename PTR>
//HOST_DEVICE void setFluxData(const CFuint f, const CFint stype, 
//			     const CFuint stateID, const CFuint cellID, 
//			     KernelData<CFreal>* kd, FluxData<PHYS>* fd,
//			     PTR cellFaces)
//{
//  fd->setStateID(RIGHT, stateID);
//  CFreal* statePtrR = (stype > 0) ? &kd->states[stateID*PHYS::NBEQS] : &kd->ghostStates[stateID*PHYS::NBEQS];  
//  CFreal* nodePtrR = (stype > 0) ? &kd->centerNodes[stateID*PHYS::DIM] : &kd->ghostNodes[stateID*PHYS::DIM];  
//  setState<PHYS>(fd->getState(RIGHT), statePtrR, fd->getNode(RIGHT), nodePtrR);
//  
//  fd->setIsBFace(stype < 0);
//  fd->setStateID(LEFT, cellID);
//  const CFuint faceID = cellFaces[f*kd->nbCells + cellID];
//  fd->setIsOutward(kd->isOutward[faceID] == cellID);
//  
//  CFreal* statePtrL = &kd->states[cellID*PHYS::NBEQS];
//  CFreal* nodePtrL = &kd->centerNodes[cellID*PHYS::DIM];
//  setState<PHYS>(fd->getState(LEFT), statePtrL, fd->getNode(LEFT), nodePtrL);
//  setFaceNormal<PHYS>(fd, &kd->normals[faceID*PHYS::DIM]);
//}

//////////////////////////////////////////////////////////////////////////////

template <typename T, CFuint SIZE>
void print(const std::string& name, T* array) 
{
  CFLog(INFO, name << " = \t");
  for (CFuint i = 0; i < SIZE; ++i) {
    CFLog(INFO, array[i] << " ");
  }
  CFLog(INFO, "\n");
}

//////////////////////////////////////////////////////////////////////////////

//template <typename MODEL>
//HOST_DEVICE void computeFaceCentroid(const CellData::Itr* cell, const CFuint faceIdx, 
//				     const CFreal* nodes, CFreal* midFaceCoord)
//{  
//  CudaEnv::CFVecSlice<CFreal, MODEL::DIM> coord(midFaceCoord);
//  coord = 0.;
//  const CFuint nbFaceNodes = cell->getNbFaceNodes(faceIdx);
//  const CFreal ovNbFaceNodes = 1./(static_cast<CFreal>(nbFaceNodes));
//  for (CFuint n = 0; n < nbFaceNodes; ++n) {
//    const CFuint cellNodeID = cell->getNodeID(faceIdx, n);
//    const CFuint nodeID = cell->getNodeID(faceIdx,n);
//    const CFreal* faceNode = &nodes[nodeID*MODEL::DIM];
//    for (CFuint d = 0; d < MODEL::DIM; ++d) {
//      coord[d] += faceNode[d];
//    }
//  }
//  coord *= ovNbFaceNodes;
//}

//////////////////////////////////////////////////////////////////////////////

//template <typename PHYS, typename POLYREC>
//__global__ void computeGradientsKernel(typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
//				       const CFuint nbCells,
//				       CFreal* states, 
//				       CFreal* nodes,
//				       CFreal* centerNodes,
//				       CFreal* ghostStates,
//				       CFreal* ghostNodes,
//				       CFreal* uX,
//				       CFreal* uY,
//				       CFreal* uZ,
//				       CFreal* limiter,
//				       CFreal* updateCoeff, 
//				       CFreal* rhs,
//				       CFreal* normals,
//				       CFint* isOutward,
//				       const CFuint* cellInfo,
//				       const CFuint* cellStencil,
//				       const CFuint* cellFaces,
//				       const CFuint* cellNodes,
//				       const CFint*  neighborTypes,
//				       const Framework::CellConn* cellConn)
//{    
//  // each thread takes care of computing the gradient for one single cell
//  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
//  
//  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
//  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
//  // dcor->init(gdcor);
//  
//  if (cellID < nbCells) {    
//    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
//			   rhs, normals, uX, uY, uZ, isOutward);
//    
//    // compute and store cell gradients at once 
//    POLYREC polyRec(dcor);
//    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
//    CellData::Itr cell = cells.getItr(cellID);
//    polyRec.computeGradients(&states[cellID*PHYS::NBEQS], &centerNodes[cellID*PHYS::DIM], &kd, &cell);
//  }
//}
      
//////////////////////////////////////////////////////////////////////////////

//template <typename PHYS, typename POLYREC, typename LIMITER>
//__global__ void computeLimiterKernel(typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol,
//				     typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
//				     const CFuint nbCells,
//				     CFreal* states, 
//				     CFreal* nodes,
//				     CFreal* centerNodes,
//				     CFreal* ghostStates,
//				     CFreal* ghostNodes,
//				     CFreal* uX,
//				     CFreal* uY,
//				     CFreal* uZ,
//				     CFreal* limiter,
//				     CFreal* updateCoeff, 
//				     CFreal* rhs,
//				     CFreal* normals,
//				     CFint* isOutward,
//				     const CFuint* cellInfo,
//				     const CFuint* cellStencil,
//				     const CFuint* cellFaces,
//				     const CFuint* cellNodes,
//				     const CFint*  neighborTypes,
//				     const Framework::CellConn* cellConn)
//{    
//  // each thread takes care of computing the gradient for one single cell
//  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
//  
//  // __shared__ typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE> s_dcol[32];
//  // typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol = &s_dcol[threadIdx.x];
//  // dcol->init(gdcol);
//  
//  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
//  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
//  // dcor->init(gdcor);
//  
//  if (cellID < nbCells) {    
//    // compute all cell quadrature points at once (size of this array is overestimated)
//    CFreal midFaceCoord[PHYS::DIM*PHYS::DIM*2];
//    
//    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
//    CellData::Itr cell = cells.getItr(cellID);
//    const CFuint nbFacesInCell = cell.getNbFacesInCell();
//    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
//      computeFaceCentroid<PHYS>(&cell, f, nodes, &midFaceCoord[f*PHYS::DIM]);
//    }
//    
//    // compute cell-based limiter at once
//    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
//			   rhs, normals, uX, uY, uZ, isOutward);
//    LIMITER limt(dcol);
//    
//    if (dcor->currRes > dcor->limitRes && (dcor->limitIter > 0 && dcor->currIter < dcor->limitIter)) {	
//      limt.limit(&kd, &cell, &midFaceCoord[0], &limiter[cellID*PHYS::NBEQS]);
//    }
//    else {
//      if (!dcor->freezeLimiter) {
//	// historical modification of the limiter
//	CudaEnv::CFVec<CFreal,PHYS::NBEQS> tmpLimiter;
//	limt.limit(&kd, &cell, &midFaceCoord[0], &tmpLimiter[0]);
//	CFuint currID = cellID*PHYS::NBEQS;
//	for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar, ++currID) {
//	  limiter[currID] = min(tmpLimiter[iVar],limiter[currID]);
//	}
//      }
//    }
//  }
//}
  
//////////////////////////////////////////////////////////////////////////////
             
template <typename SCHEME>
__global__ void computeFluxKernel(typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
				  const CFuint nbCells,
				  CFreal* states, 
				  CFreal* updateCoeff, 
				  CFreal* rhs,
				  const CFuint* cellInfo,
				  const CFuint* cellStencil,
				  const CFuint* cellFaces,
				  const CFuint* cellNodes,
				  const CFint*  neighborTypes,
				  const Framework::CellConn* cellConn)
{    
  // each thread takes care of computing the flux for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  // __shared__ typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE> s_dcof[32];
  // typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof = &s_dcof[threadIdx.x];
  // dcof->init(gdcof);
  
  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
  // dcor->init(gdcor);
  
  // __shared__ typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE> s_dcop[32];
  // typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop = &s_dcop[threadIdx.x];
  // dcop->init(gdcop);
  
  if (cellID < nbCells) {
    // reset the rhs and update coefficients to 0
    CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> res(&rhs[cellID*SCHEME::MODEL::NBEQS]);
    res = 0.;
    updateCoeff[cellID] = 0.;
    
//    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
//			   rhs, normals, uX, uY, uZ, isOutward);
//    
//    // compute and store cell gradients at once 
//    SCHEME fluxScheme(dcof);
//    CFreal midFaceCoord[SCHEME::MODEL::DIM*SCHEME::MODEL::DIM*2];
//    FluxData<typename SCHEME::MODEL> currFd; currFd.initialize();
//    typename SCHEME::MODEL pmodel(dcop);
//    
//    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
//    CellData::Itr cell = cells.getItr(cellID);
//    
//    // compute the fluxes
//    const CFuint nbFacesInCell = cell.getNbActiveFacesInCell();
//    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
//      const CFint stype = cell.getNeighborType(f);
//      
//      if (stype != 0) { // skip all partition faces
//	// set all flux data for the current face
//	const CFuint stateID = cell.getNeighborID(f);
//	setFluxData(f, stype, stateID, cellID, &kd, &currFd, cellFaces);
//	
//	// compute face quadrature points (centroid)
//	CFreal* faceCenters = &midFaceCoord[f*SCHEME::MODEL::DIM];
//	computeFaceCentroid<typename SCHEME::MODEL>(&cell, f, nodes, faceCenters);
//	
//	// extrapolate solution on quadrature points on both sides of the face
//	polyRec.extrapolateOnFace(&currFd, faceCenters, uX, uY, uZ, limiter);
//	
//	// compute the convective flux across the face
//        fluxScheme.prepareComputation(&currFd, &pmodel);
//	fluxScheme(&currFd, &pmodel);
//	
//	// update the residual
//	CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> ress(currFd.getResidual());
//	res -= ress;
//	
//	// update the update coefficient
//	updateCoeff[cellID] += currFd.getUpdateCoeff();
//      }
//    }
//
//    // get the states in this cell
//    m_cellStates = m_cell->getStates();
//      
//    // set the cell data
//    setCellData();
//      
//    // compute the divergence of the discontinuous flux (-divFD+divhFD)
//    computeDivDiscontFlx(m_divContFlx);
//      
//    // update RHS
//    updateRHS();
     
  }
}
  
//////////////////////////////////////////////////////////////////////////////

//template <typename SCHEME, typename POLYREC, typename LIMITER>
//void computeFluxCPU(CFuint nbThreadsOMP,
//		    typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof,
//		    typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
//		    typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol,
//		    typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
//		    const CFuint nbCells,
//		    CFreal* states, 
//		    CFreal* nodes,
//		    CFreal* centerNodes,
//		    CFreal* ghostStates,
//		    CFreal* ghostNodes,
//		    CFreal* uX,
//		    CFreal* uY,
//		    CFreal* uZ,
//		    CFreal* limiter,
//		    CFreal* updateCoeff, 
//		    CFreal* rhs,
//		    CFreal* normals,
//		    CFint* isOutward,
//		    const CFuint* cellInfo,
//		    const CFuint* cellStencil,
//		    const CFuint* cellFaces,
//		    const CFuint* cellNodes,
//		    const CFint* neighborTypes,
//		    const Framework::CellConn* cellConn)
//{ 
//  typedef typename SCHEME::MODEL PHYS;
//  
//  FluxData<PHYS> fd;
//#ifndef CF_HAVE_OMP  
//  fd.initialize();
//  FluxData<PHYS>* currFd = &fd;
//  cf_assert(currFd != CFNULL);
//#endif
//  POLYREC polyRec(dcor);
//  SCHEME fluxScheme(dcof);
//  LIMITER limt(dcol);
//  PHYS pmodel(dcop);
//  
//  CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
//  KernelData<CFreal> kd(nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
//			rhs, normals, uX, uY, uZ, isOutward);
//  
//  CFreal midFaceCoord[PHYS::DIM*PHYS::DIM*2];
//  CudaEnv::CFVec<CFreal,PHYS::NBEQS> tmpLimiter;
//
//#ifdef CF_HAVE_OMP
//  //const CFuint nThr = omp_get_num_procs();
//  // omp_set_num_threads(nbThreadsOMP);
//#pragma omp num_thread(nbThreadsOMP) parallel private(polyRec) private(fd)
//{
//  #pragma omp for
//#endif 
//  // compute the cell-based gradients
//  for (CFuint cellID = 0; cellID < nbCells; ++cellID) {
//#ifdef CF_HAVE_OMP
//    fd.initialize();
//    FluxData<PHYS>* currFd = &fd;
//    cf_assert(currFd != CFNULL);
//#endif 
//    CellData::Itr cell = cells.getItr(cellID);
//    polyRec.computeGradients(&states[cellID*PHYS::NBEQS], &centerNodes[cellID*PHYS::DIM], &kd, &cell);
//  }
//#ifdef CF_HAVE_OMP
//}
//#endif
//
//#ifdef CF_HAVE_OMP  
//#pragma omp num_thread(nbThreadsOMP) parallel private(limt) private(kd)
//{
//  #pragma omp for
//#endif 
//  // compute the cell based limiter 
//  for (CFuint cellID = 0; cellID < nbCells; ++cellID) {
//  // for (CellData::Itr cell = cells.begin(); cell <= cells.end(); ++cell) {
//    CellData::Itr cell = cells.getItr(cellID);
//    // compute all cell quadrature points at once (size of this array is overestimated)
//    const CFuint nbFacesInCell = cell.getNbFacesInCell();
//    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
//      computeFaceCentroid<PHYS>(&cell, f, nodes, &midFaceCoord[f*PHYS::DIM]);
//    }
//    
//    //   const CFuint cellID = cell.getCellID();
//    if (dcor->currRes > dcor->limitRes && (dcor->limitIter > 0 && dcor->currIter < dcor->limitIter)) {	
//      // compute cell-based limiter
//      limt.limit(&kd, &cell, &midFaceCoord[0], &limiter[cellID*PHYS::NBEQS]);
//    }
//    else {
//      if (!dcor->freezeLimiter) {
//	// historical modification of the limiter
//	limt.limit(&kd, &cell, &midFaceCoord[0], &tmpLimiter[0]);
//	CFuint currID = cellID*PHYS::NBEQS;
//	for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar, ++currID) {
//	  limiter[currID] = min(tmpLimiter[iVar],limiter[currID]);
//	}
//      }
//    }
//  }
//#ifdef CF_HAVE_OMP
//}
//
//#pragma omp num_thread(nbThreadsOMP) parallel private(fd) private(kd) private(fluxScheme) private(pmodel)
//{
//  #pragma omp for
//#endif 
//  // compute the fluxes
//  for (CFuint cellID = 0; cellID < nbCells; ++cellID) {
//  //  for (CellData::Itr cell = cells.begin(); cell <= cells.end(); ++cell) {
//#ifdef CF_HAVE_OMP
//    fd.initialize();
//    FluxData<PHYS>* currFd = &fd;
//    cf_assert(currFd != CFNULL);
//#endif
//    // reset the rhs and update coefficients to 0
//   // const CFuint cellID = cell.getCellID();
//    CudaEnv::CFVecSlice<CFreal,PHYS::NBEQS> res(&rhs[cellID*PHYS::NBEQS]);
//    res = 0.;
//    updateCoeff[cellID] = 0.;
//
//    CellData::Itr cell = cells.getItr(cellID);   
//    const CFuint nbFacesInCell = cell.getNbActiveFacesInCell();
//    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
//      const CFint stype = cell.getNeighborType(f);
//      
//      if (stype != 0) { // skip all partition faces
//	const CFuint stateID =  cell.getNeighborID(f);
//	setFluxData(f, stype, stateID, cellID, &kd, currFd, cellFaces);
//	
//	// compute face quadrature points (centroid)
//	CFreal* faceCenters = &midFaceCoord[f*PHYS::DIM];
//	computeFaceCentroid<PHYS>(&cell, f, nodes, faceCenters);
//	
//	// extrapolate solution on quadrature points on both sides of the face
//	polyRec.extrapolateOnFace(currFd, faceCenters, uX, uY, uZ, limiter);
//        fluxScheme.prepareComputation(currFd, &pmodel);
//	fluxScheme(currFd, &pmodel); // compute the convective flux across the face
//	
//	for (CFuint iEq = 0; iEq < PHYS::NBEQS; ++iEq) {
//	  const CFreal value = currFd->getResidual()[iEq];
//	  res[iEq] -= value;  // update the residual 
//	}
//	
//	// update the update coefficient
//	updateCoeff[cellID] += currFd->getUpdateCoeff();
//      }
//    }
//  }
//#ifdef CF_HAVE_OMP
//} 
//#endif
//}

//////////////////////////////////////////////////////////////////////////////

template <typename SCHEME, typename PHYSICS, CFuint NB_BLOCK_THREADS>
void ConvRHSFluxReconstructionCUDA<SCHEME,PHYSICS,NB_BLOCK_THREADS>::execute()
{
  using namespace COOLFluiD::Framework;
  using namespace COOLFluiD::Common;
  
  CFTRACEBEGIN;
  
  CFLog(VERBOSE, "ConvRHSFluxReconstructionCUDA::execute() START\n");
  
  initializeComputationRHS();

  const CFuint nbCells = socket_states.getDataHandle().size();
  cf_assert(nbCells > 0);
  DataHandle<CFreal> updateCoeff = socket_updateCoeff.getDataHandle();
  DataHandle<CFreal> rhs = socket_rhs.getDataHandle(); 
  
  SafePtr<SCHEME> lf  = getMethodData().getRiemannFlux().d_castTo<SCHEME>();
  SafePtr<typename PHYSICS::PTERM> phys = PhysicalModelStack::getActive()->getImplementor()->
    getConvectiveTerm().d_castTo<typename PHYSICS::PTERM>();
  
#ifdef CF_HAVE_CUDA
  typedef typename SCHEME::template DeviceFunc<GPU, PHYSICS> FluxScheme;  
#else
  typedef typename SCHEME::template DeviceFunc<CPU, PHYSICS> FluxScheme;
#endif 
  
  if (m_onGPU) {
#ifdef CF_HAVE_CUDA

    CudaEnv::CudaTimer& timer = CudaEnv::CudaTimer::getInstance();
    timer.start();
    
    // copy of data that change at every iteration
    socket_states.getDataHandle().getGlobalArray()->put(); 
     
    CFLog(VERBOSE, "ConvRHSFluxReconstructionCUDA::execute() => CPU-->GPU data transfer took " << timer.elapsed() << " s\n");
    timer.start();
    
    ConfigOptionPtr<SCHEME,  NOTYPE, GPU> dcof(lf);
    ConfigOptionPtr<typename PHYSICS::PTERM, NOTYPE, GPU> dcop(phys);
    
    const CFuint blocksPerGrid = CudaEnv::CudaDeviceManager::getInstance().getBlocksPerGrid(nbCells);
    const CFuint nThreads = CudaEnv::CudaDeviceManager::getInstance().getNThreads();
    
    //dim3 blocks(m_nbBlocksPerGridX, m_nbBlocksPerGridY);
    
    //hipFuncSetCacheConfig(reinterpret_cast<const void*>("computeGradientsKernel"), hipFuncCachePreferL1);
    
    // hipFuncSetCacheConfig(reinterpret_cast<const void*>("computeFluxKernel"), hipFuncCachePreferL1);
    
    // compute the convective flux in each cell
    computeFluxKernel<FluxScheme> <<<blocksPerGrid,nThreads>>> 
      (dcop.getPtr(),
       nbCells,
       socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
       updateCoeff.getLocalArray()->ptrDev(), 
       rhs.getLocalArray()->ptrDev(),
       m_cellInfo.ptrDev(),
       m_cellStencil.ptrDev(),
       m_cellFaces->getPtr()->ptrDev(),
       m_neighborTypes.ptrDev(),
       m_cellConn.ptrDev());
    
    CFLog(VERBOSE, "ConvRHSFluxReconstructionCUDA::execute() => computeFluxKernel took " << timer.elapsed() << " s\n");
    
    timer.start();
    rhs.getLocalArray()->get();
    updateCoeff.getLocalArray()->get();
    CFLog(VERBOSE, "ConvRHSFluxReconstructionCUDA::execute() => GPU-->CPU data transfer took " << timer.elapsed() << " s\n");

#endif
}
  else {
    // AL: useful fo debugging
    // for (CFuint i = 0; i <  m_ghostStates.size()/9; ++i) {
    //   std::cout.precision(12); std::cout << "g" << i << " => ";
    //   for (CFuint j = 0; j < 9; ++j) {
    // 	std::cout << m_ghostStates[i*9+j] << " ";
    //   }
    //   std::cout << "\n";
    // }
    // for (CFuint i = 0; i <  socket_states.getDataHandle().size(); ++i) {
    //   std::cout.precision(12); std::cout << i << " => "<< *socket_states.getDataHandle()[i] <<"\n";
    // }
    
//    ConfigOptionPtr<SCHEME>  dcof(lf);
//    ConfigOptionPtr<POLYREC> dcor(pr);
//    ConfigOptionPtr<LIMITER> dcol(lm);
//    ConfigOptionPtr<typename PHYSICS::PTERM> dcop(phys);
//    
//    computeFluxCPU<FluxScheme, PolyRec, Limiter>
//      (m_nbThreadsOMP,
//       dcof.getPtr(),
//       dcor.getPtr(),
//       dcol.getPtr(),
//       dcop.getPtr(),
//       nbCells,
//       socket_states.getDataHandle().getGlobalArray()->ptr(), 
//       socket_nodes.getDataHandle().getGlobalArray()->ptr(),
//       m_centerNodes.ptr(), 
//       m_ghostStates.ptr(),
//       m_ghostNodes.ptr(),
//       socket_uX.getDataHandle().getLocalArray()->ptr(),
//       socket_uY.getDataHandle().getLocalArray()->ptr(),
//       socket_uZ.getDataHandle().getLocalArray()->ptr(),
//       socket_limiter.getDataHandle().getLocalArray()->ptr(),
//       updateCoeff.getLocalArray()->ptr(), 
//       rhs.getLocalArray()->ptr(),
//       normals.getLocalArray()->ptr(),
//       isOutward.getLocalArray()->ptr(),
//       m_cellInfo.ptr(),
//       m_cellStencil.ptr(),
//       m_cellFaces->getPtr()->ptr(),
//       m_cellNodes->getPtr()->ptr(),
//       m_neighborTypes.ptr(),
//       m_cellConn.ptr());
  }
  
// for (int i = 0; i < updateCoeff.size(); ++i) {
//      std::cout << "updateCoeff[" << i << "] = " << updateCoeff[i]  << std::endl;
//       /* std::cout << "rhs[" << i << "] = ";
//        for (int j = 0; j < 9; ++j) {
//          std::cout << rhs[i*9+j] << " ";
//        }
//        std::cout << std::endl;*/
// } 
//   abort();
  // for (;;) {}
  
  //finalizeComputationRHS();
  
  CFLog(VERBOSE, "ConvRHSFluxReconstructionCUDA::execute() END\n");
  
  CFTRACEEND;
}

//////////////////////////////////////////////////////////////////////////////

    } // namespace FluxReconstructionMethod

} // namespace COOLFluiD
