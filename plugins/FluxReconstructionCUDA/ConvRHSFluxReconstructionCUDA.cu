#include "hip/hip_runtime.h"
#include "FluxReconstructionCUDA/ConvRHSFluxReconstructionCUDA.hh"
#include "Framework/MeshData.hh"
#include "Framework/CellConn.hh"
#include "Config/ConfigOptionPtr.hh"
#include "Framework/CudaDeviceManager.hh"
#include "Common/CUDA/CFVec.hh"
#include "Framework/CudaTimer.hh"

#include "FluxReconstructionMethod/FluxData.hh"
#include "FluxReconstructionMethod/KernelData.hh"
#include "FluxReconstructionMethod/CellData.hh"

#include "FluxReconstructionCUDA/FluxReconstructionCUDA.hh"
#include "Framework/MethodCommandProvider.hh"
#include "Framework/VarSetListT.hh"
#include "NavierStokes/Euler2DVarSetT.hh"
#include "NavierStokes/Euler2DConsT.hh"

#include "FluxReconstructionMethod/LaxFriedrichsFlux.hh"
#include <stdio.h>

//////////////////////////////////////////////////////////////////////////////

using namespace COOLFluiD::Framework;
using namespace COOLFluiD::Common;
using namespace COOLFluiD::Config;
using namespace COOLFluiD::Physics::NavierStokes;

//////////////////////////////////////////////////////////////////////////////

namespace COOLFluiD {

    namespace FluxReconstructionMethod {

//////////////////////////////////////////////////////////////////////////////

#define FR_EULER_RHS_PROV(__dim__,__svars__,__uvars__,__nbBThreads__,__providerName__) \
MethodCommandProvider<ConvRHSFluxReconstructionCUDA<LaxFriedrichsFlux, \
                      VarSetListT<Euler##__dim__##__svars__##T, Euler##__dim__##__uvars__##T>, __nbBThreads__>, \
		      FluxReconstructionSolverData,FluxReconstructionCUDAModule>	\
FR_RhsEuler##__dim__##__svars__##__uvars__##__nbBThreads__##Provider(__providerName__);
// 48 block threads (default)
FR_EULER_RHS_PROV(2D, Cons, Cons, 48, "EulerFRLaxFriedrichs2DCons")
//FR_EULER_RHS_PROV(3D, Cons, Cons, 48, "EulerFRLaxFried3DCons")
//FR_NS_RHS_PROV(2D, ProjectionCons, ProjectionPrim, 48, "CellLaxFriedMHD2DPrim")
//FR_NS_RHS_PROV(3D, ProjectionCons, ProjectionPrim, 48, "CellLaxFriedMHD3DPrim")
#undef FR_EULER_RHS_PROV

//////////////////////////////////////////////////////////////////////////////

template <typename PHYS>
HOST_DEVICE inline void setState(CFreal* state, CFreal* statePtr)
{
  // copy the state node data to shared memory
  //for (CFuint i = 0; i < PHYS::DIM; ++i) {node[i] = nodePtr[i];}
  // copy the state data to shared memory
  for (CFuint i = 0; i < PHYS::NBEQS; ++i) {state[i] = statePtr[i];} 
}
      
//////////////////////////////////////////////////////////////////////////////

template <typename PHYS>
HOST_DEVICE void setFluxData(const CFuint stateID, const CFuint cellID, 
			     KernelData<CFreal>* kd, FluxData<PHYS>* fd, const CFuint iSol)
{
  fd->setStateID(LEFT, stateID);
  CFreal* statePtrR = &kd->states[stateID*PHYS::NBEQS];  

  setState<PHYS>(fd->getState(iSol), statePtrR);

  fd->setNbSolPnts(kd->nbSolPnts);
}

//////////////////////////////////////////////////////////////////////////////

template <typename T, CFuint SIZE>
void print(const std::string& name, T* array) 
{
  CFLog(INFO, name << " = \t");
  for (CFuint i = 0; i < SIZE; ++i) {
    CFLog(INFO, array[i] << " ");
  }
  CFLog(INFO, "\n");
}

//////////////////////////////////////////////////////////////////////////////

//template <typename MODEL>
//HOST_DEVICE void computeFaceCentroid(const CellData::Itr* cell, const CFuint faceIdx, 
//				     const CFreal* nodes, CFreal* midFaceCoord)
//{  
//  CudaEnv::CFVecSlice<CFreal, MODEL::DIM> coord(midFaceCoord);
//  coord = 0.;
//  const CFuint nbFaceNodes = cell->getNbFaceNodes(faceIdx);
//  const CFreal ovNbFaceNodes = 1./(static_cast<CFreal>(nbFaceNodes));
//  for (CFuint n = 0; n < nbFaceNodes; ++n) {
//    const CFuint cellNodeID = cell->getNodeID(faceIdx, n);
//    const CFuint nodeID = cell->getNodeID(faceIdx,n);
//    const CFreal* faceNode = &nodes[nodeID*MODEL::DIM];
//    for (CFuint d = 0; d < MODEL::DIM; ++d) {
//      coord[d] += faceNode[d];
//    }
//  }
//  coord *= ovNbFaceNodes;
//}

//////////////////////////////////////////////////////////////////////////////

//template <typename PHYS, typename POLYREC>
//__global__ void computeGradientsKernel(typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
//				       const CFuint nbCells,
//				       CFreal* states, 
//				       CFreal* nodes,
//				       CFreal* centerNodes,
//				       CFreal* ghostStates,
//				       CFreal* ghostNodes,
//				       CFreal* uX,
//				       CFreal* uY,
//				       CFreal* uZ,
//				       CFreal* limiter,
//				       CFreal* updateCoeff, 
//				       CFreal* rhs,
//				       CFreal* normals,
//				       CFint* isOutward,
//				       const CFuint* cellInfo,
//				       const CFuint* cellStencil,
//				       const CFuint* cellFaces,
//				       const CFuint* cellNodes,
//				       const CFint*  neighborTypes,
//				       const Framework::CellConn* cellConn)
//{    
//  // each thread takes care of computing the gradient for one single cell
//  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
//  
//  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
//  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
//  // dcor->init(gdcor);
//  
//  if (cellID < nbCells) {    
//    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
//			   rhs, normals, uX, uY, uZ, isOutward);
//    
//    // compute and store cell gradients at once 
//    POLYREC polyRec(dcor);
//    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
//    CellData::Itr cell = cells.getItr(cellID);
//    polyRec.computeGradients(&states[cellID*PHYS::NBEQS], &centerNodes[cellID*PHYS::DIM], &kd, &cell);
//  }
//}
      
//////////////////////////////////////////////////////////////////////////////

//template <typename PHYS, typename POLYREC, typename LIMITER>
//__global__ void computeLimiterKernel(typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol,
//				     typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
//				     const CFuint nbCells,
//				     CFreal* states, 
//				     CFreal* nodes,
//				     CFreal* centerNodes,
//				     CFreal* ghostStates,
//				     CFreal* ghostNodes,
//				     CFreal* uX,
//				     CFreal* uY,
//				     CFreal* uZ,
//				     CFreal* limiter,
//				     CFreal* updateCoeff, 
//				     CFreal* rhs,
//				     CFreal* normals,
//				     CFint* isOutward,
//				     const CFuint* cellInfo,
//				     const CFuint* cellStencil,
//				     const CFuint* cellFaces,
//				     const CFuint* cellNodes,
//				     const CFint*  neighborTypes,
//				     const Framework::CellConn* cellConn)
//{    
//  // each thread takes care of computing the gradient for one single cell
//  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
//  
//  // __shared__ typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE> s_dcol[32];
//  // typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol = &s_dcol[threadIdx.x];
//  // dcol->init(gdcol);
//  
//  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
//  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
//  // dcor->init(gdcor);
//  
//  if (cellID < nbCells) {    
//    // compute all cell quadrature points at once (size of this array is overestimated)
//    CFreal midFaceCoord[PHYS::DIM*PHYS::DIM*2];
//    
//    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
//    CellData::Itr cell = cells.getItr(cellID);
//    const CFuint nbFacesInCell = cell.getNbFacesInCell();
//    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
//      computeFaceCentroid<PHYS>(&cell, f, nodes, &midFaceCoord[f*PHYS::DIM]);
//    }
//    
//    // compute cell-based limiter at once
//    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
//			   rhs, normals, uX, uY, uZ, isOutward);
//    LIMITER limt(dcol);
//    
//    if (dcor->currRes > dcor->limitRes && (dcor->limitIter > 0 && dcor->currIter < dcor->limitIter)) {	
//      limt.limit(&kd, &cell, &midFaceCoord[0], &limiter[cellID*PHYS::NBEQS]);
//    }
//    else {
//      if (!dcor->freezeLimiter) {
//	// historical modification of the limiter
//	CudaEnv::CFVec<CFreal,PHYS::NBEQS> tmpLimiter;
//	limt.limit(&kd, &cell, &midFaceCoord[0], &tmpLimiter[0]);
//	CFuint currID = cellID*PHYS::NBEQS;
//	for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar, ++currID) {
//	  limiter[currID] = min(tmpLimiter[iVar],limiter[currID]);
//	}
//      }
//    }
//  }
//}
  
//////////////////////////////////////////////////////////////////////////////

template <typename SCHEME, typename PHYS>
__global__ void computeStateLocalRHSKernel(typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof,
                                  typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
                                  const CFuint nbCells,
				  CFreal* states, 
                                  CFreal* updateCoeff, 
				  CFreal* rhs,
                                  CFreal* solPntNormals,
                                  const CFuint nbSolPnts,
                                  const CFuint nbrFaces,
				  const CFuint* cellInfo,
                                  const CFuint* stateIDs,
                                  const CFuint* neighbCellIDs,
                                  const CFuint dim,
                                  const CFuint nbrEqs,
                                  const CFuint nbrFlxPnts,
                                  const CFuint nbrSolSolDep,
                                  const CFuint* solSolDep,
                                  const CFuint nbrSolFlxDep,
                                  const CFuint* solFlxDep,
                                  const CFreal* solPolyDerivAtSolPnts,
                                  const CFreal* solPolyValsAtFlxPnts,
                                  const CFuint* flxPntFlxDim,
                                  const CFreal* corrFctDiv)
{    
  // one thread per cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;

  // __shared__ typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE> s_dcof[32];
  // typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof = &s_dcof[threadIdx.x];
  // dcof->init(gdcof);
  
  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
  // dcor->init(gdcor);
  
  // __shared__ typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE> s_dcop[32];
  // typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop = &s_dcop[threadIdx.x];
  // dcop->init(gdcop);
  
  if (cellID < nbCells) 
  { 
    // current kernel data
    KernelData<CFreal> kd (nbCells, states, updateCoeff, rhs, solPntNormals, nbSolPnts);

    // current flux data
    FluxData<typename SCHEME::MODEL> currFd; 
    
    // initialize flux data
    currFd.initialize();
    
    // physical model
    typename SCHEME::MODEL pmodel(dcop);
    SCHEME fluxScheme(dcof);
    
    // current cell data
    CellData cells(nbCells, cellInfo, stateIDs, neighbCellIDs, nbrFaces, nbSolPnts);
    
    // get current cell
    CellData::Itr cell = cells.getItr(cellID);
          
    const CFuint nbFlxPntFlx = SCHEME::MODEL::NBEQS*8;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx> flxPntFlx;
    
    flxPntFlx = 0.0;

    // loop over sol pnts to compute flux
    for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
    {
      // get current state ID
      const CFuint stateID = cell.getStateID(iSolPnt);
      //printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
    
    if (cellID == 0) printf("GPUstate: %f %f %f %f\n", kd.states[0], kd.states[1], kd.states[2], kd.states[3]);
    
      setFluxData(stateID, cellID, &kd, &currFd, iSolPnt);

      const CFuint nbNormals = PHYS::DIM*PHYS::DIM;

      CudaEnv::CFVecSlice<CFreal,nbNormals> n(&(kd.solPntNormals[stateID*nbNormals]));

      CudaEnv::CFVecSlice<CFreal,nbNormals> nFd(currFd.getScaledNormal(iSolPnt));
      
      for (CFuint i = 0; i < nbNormals; ++i) 
      {
        nFd[i] = n[i];
      }
      
      // get the flux
      fluxScheme.prepareComputation(&currFd, &pmodel);
      
      fluxScheme(&currFd, &pmodel, false, iSolPnt);
      
      // loop over sol pnts to compute flux
      for (CFuint iDim = 0; iDim < dim; ++iDim)
      {
        if (cellID == 0) printf("HERE4 iSol: %d, iDim: %d, flux: %f %f %f %f \n", iSolPnt, iDim, currFd.getFlux(iSolPnt, iDim)[0], currFd.getFlux(iSolPnt, iDim)[1], currFd.getFlux(iSolPnt, iDim)[2], currFd.getFlux(iSolPnt, iDim)[3]);
      }
      
      // Loop over solution pnts to count the factor of all sol pnt polys
      for (CFuint jSolPnt = 0; jSolPnt < nbrSolSolDep; ++jSolPnt)
      { 
      
        const CFuint jSolIdx = solSolDep[iSolPnt*nbrSolSolDep+jSolPnt]; //(*m_solSolDep)[iSolPnt][jSolPnt];
        
        // get current vector slice out of rhs
        CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> res(&rhs[stateID*SCHEME::MODEL::NBEQS]);

        // Loop over deriv directions and sum them to compute divergence
        for (CFuint iDir = 0; iDir < dim; ++iDir)
        {
          const CFreal polyCoef = solPolyDerivAtSolPnts[iSolPnt*dim*nbSolPnts+iDir*nbSolPnts+jSolIdx];//(*m_solPolyDerivAtSolPnts)[jSolPnt][iDir][iSolIdx]; 
          
          if (cellID == 0) printf("polyCoef: %f\n", polyCoef);
          
          // Loop over conservative fluxes 
          for (CFuint iEq = 0; iEq < nbrEqs; ++iEq)
          {
            // Store divFD in the vector that will be divFC
            res[iEq] -= polyCoef*(currFd.getFlux(iSolPnt, iDir)[iEq]);
            if (cellID == 0) printf("res %f \n", res[iEq]);
	  }
        }
      }
      
      // extrapolate the fluxes to the flux points
      for (CFuint iFlxPnt = 0; iFlxPnt < nbrSolFlxDep; ++iFlxPnt)
      {
        const CFuint flxIdx = solFlxDep[iSolPnt*nbrSolFlxDep+iFlxPnt];
        const CFuint dim = flxPntFlxDim[flxIdx];
        // Loop over conservative fluxes 
        for (CFuint iEq = 0; iEq < nbrEqs; ++iEq)
        {
          flxPntFlx[flxIdx*nbrEqs+iEq] += solPolyValsAtFlxPnts[flxIdx*nbrFlxPnts+iSolPnt]*currFd.getFlux(iSolPnt, dim)[iEq];
        }
      }
    }
    
    for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
    {
      // get current state ID
      const CFuint stateID = cell.getStateID(iSolPnt);
      
      // get current vector slice out of rhs
      CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> res(&rhs[stateID*SCHEME::MODEL::NBEQS]);
        
      // add divhFD to the residual updates
      for (CFuint iFlxPnt = 0; iFlxPnt < nbrSolFlxDep; ++iFlxPnt)
      {
        const CFuint flxIdx = solFlxDep[iSolPnt*nbrSolFlxDep+iFlxPnt];

        // get the divergence of the correction function
        const CFreal divh = corrFctDiv[iSolPnt*nbrFlxPnts+iFlxPnt];
 
        // Fill in the corrections
        for (CFuint iVar = 0; iVar < nbrEqs; ++iVar)
        {
          res[iVar] += flxPntFlx[flxIdx*nbrEqs+iVar] * divh; 
        }
      }
    }
    
    
    for (CFuint iFace = 0; iFace < nbrFaces; ++iFace)
    {
      // reset flx pnt fluxes  
      flxPntFlx = 0.0;
        
      // loop over sol pnts to compute flux
      for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
      {
        // get current state ID
        const CFuint neighbStateID = cell.getNeighbStateID(iFace,iSolPnt);
        
        const CFuint neighbCellID = cell.getNeighbCellID(iFace);
    
        setFluxData(neighbStateID, neighbCellID, &kd, &currFd, iSolPnt);

        const CFuint nbNormals = PHYS::DIM*PHYS::DIM;

        CudaEnv::CFVecSlice<CFreal,nbNormals> n(&(kd.solPntNormals[neighbStateID*nbNormals]));

        CudaEnv::CFVecSlice<CFreal,nbNormals> nFd(currFd.getScaledNormal(iSolPnt));
      
        for (CFuint i = 0; i < nbNormals; ++i) 
        {
          nFd[i] = n[i];
        }
      
        // get the flux
        fluxScheme.prepareComputation(&currFd, &pmodel);
      
        fluxScheme(&currFd, &pmodel, true, iSolPnt);
      
        // extrapolate the fluxes to the flux points
        for (CFuint iFlxPnt = 0; iFlxPnt < nbrSolFlxDep; ++iFlxPnt)
        {
          const CFuint flxIdx = solFlxDep[iSolPnt*nbrSolFlxDep+iFlxPnt];
          const CFuint dim = flxPntFlxDim[flxIdx];
          // Loop over conservative fluxes 
          for (CFuint iEq = 0; iEq < nbrEqs; ++iEq)
          {
            flxPntFlx[flxIdx*nbrEqs+iEq] += solPolyValsAtFlxPnts[flxIdx*nbrFlxPnts+iSolPnt]*currFd.getFlux(iSolPnt, dim)[iEq];
          }
        }
      }
    }
      
    
    if (cellID == 0) printf("end sol pnt \n",cellID);
    
//
//  // get residual factor
//  const CFreal resFactor = getMethodData().getResFactor();
//
//  // update rhs
//  for (CFuint iState = 0; iState < m_nbrSolPnts; ++iState)
//  {
//    CFuint resID = m_nbrEqs*( (*m_cellStates)[iState]->getLocalID() );
//    for (CFuint iVar = 0; iVar < m_nbrEqs; ++iVar)
//    {
//      rhs[resID+iVar] += resFactor*m_divContFlx[iState][iVar];
//    }
//  }
    
  }
}
  
//////////////////////////////////////////////////////////////////////////////

//template <typename SCHEME, typename POLYREC, typename LIMITER>
//void computeFluxCPU(CFuint nbThreadsOMP,
//		    typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof,
//		    typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
//		    typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol,
//		    typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
//		    const CFuint nbCells,
//		    CFreal* states, 
//		    CFreal* nodes,
//		    CFreal* centerNodes,
//		    CFreal* ghostStates,
//		    CFreal* ghostNodes,
//		    CFreal* uX,
//		    CFreal* uY,
//		    CFreal* uZ,
//		    CFreal* limiter,
//		    CFreal* updateCoeff, 
//		    CFreal* rhs,
//		    CFreal* normals,
//		    CFint* isOutward,
//		    const CFuint* cellInfo,
//		    const CFuint* cellStencil,
//		    const CFuint* cellFaces,
//		    const CFuint* cellNodes,
//		    const CFint* neighborTypes,
//		    const Framework::CellConn* cellConn)
//{ 
//  typedef typename SCHEME::MODEL PHYS;
//  
//  FluxData<PHYS> fd;
//#ifndef CF_HAVE_OMP  
//  fd.initialize();
//  FluxData<PHYS>* currFd = &fd;
//  cf_assert(currFd != CFNULL);
//#endif
//  POLYREC polyRec(dcor);
//  SCHEME fluxScheme(dcof);
//  LIMITER limt(dcol);
//  PHYS pmodel(dcop);
//  
//  CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
//  KernelData<CFreal> kd(nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
//			rhs, normals, uX, uY, uZ, isOutward);
//  
//  CFreal midFaceCoord[PHYS::DIM*PHYS::DIM*2];
//  CudaEnv::CFVec<CFreal,PHYS::NBEQS> tmpLimiter;
//
//#ifdef CF_HAVE_OMP
//  //const CFuint nThr = omp_get_num_procs();
//  // omp_set_num_threads(nbThreadsOMP);
//#pragma omp num_thread(nbThreadsOMP) parallel private(polyRec) private(fd)
//{
//  #pragma omp for
//#endif 
//  // compute the cell-based gradients
//  for (CFuint cellID = 0; cellID < nbCells; ++cellID) {
//#ifdef CF_HAVE_OMP
//    fd.initialize();
//    FluxData<PHYS>* currFd = &fd;
//    cf_assert(currFd != CFNULL);
//#endif 
//    CellData::Itr cell = cells.getItr(cellID);
//    polyRec.computeGradients(&states[cellID*PHYS::NBEQS], &centerNodes[cellID*PHYS::DIM], &kd, &cell);
//  }
//#ifdef CF_HAVE_OMP
//}
//#endif
//
//#ifdef CF_HAVE_OMP  
//#pragma omp num_thread(nbThreadsOMP) parallel private(limt) private(kd)
//{
//  #pragma omp for
//#endif 
//  // compute the cell based limiter 
//  for (CFuint cellID = 0; cellID < nbCells; ++cellID) {
//  // for (CellData::Itr cell = cells.begin(); cell <= cells.end(); ++cell) {
//    CellData::Itr cell = cells.getItr(cellID);
//    // compute all cell quadrature points at once (size of this array is overestimated)
//    const CFuint nbFacesInCell = cell.getNbFacesInCell();
//    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
//      computeFaceCentroid<PHYS>(&cell, f, nodes, &midFaceCoord[f*PHYS::DIM]);
//    }
//    
//    //   const CFuint cellID = cell.getCellID();
//    if (dcor->currRes > dcor->limitRes && (dcor->limitIter > 0 && dcor->currIter < dcor->limitIter)) {	
//      // compute cell-based limiter
//      limt.limit(&kd, &cell, &midFaceCoord[0], &limiter[cellID*PHYS::NBEQS]);
//    }
//    else {
//      if (!dcor->freezeLimiter) {
//	// historical modification of the limiter
//	limt.limit(&kd, &cell, &midFaceCoord[0], &tmpLimiter[0]);
//	CFuint currID = cellID*PHYS::NBEQS;
//	for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar, ++currID) {
//	  limiter[currID] = min(tmpLimiter[iVar],limiter[currID]);
//	}
//      }
//    }
//  }
//#ifdef CF_HAVE_OMP
//}
//
//#pragma omp num_thread(nbThreadsOMP) parallel private(fd) private(kd) private(fluxScheme) private(pmodel)
//{
//  #pragma omp for
//#endif 
//  // compute the fluxes
//  for (CFuint cellID = 0; cellID < nbCells; ++cellID) {
//  //  for (CellData::Itr cell = cells.begin(); cell <= cells.end(); ++cell) {
//#ifdef CF_HAVE_OMP
//    fd.initialize();
//    FluxData<PHYS>* currFd = &fd;
//    cf_assert(currFd != CFNULL);
//#endif
//    // reset the rhs and update coefficients to 0
//   // const CFuint cellID = cell.getCellID();
//    CudaEnv::CFVecSlice<CFreal,PHYS::NBEQS> res(&rhs[cellID*PHYS::NBEQS]);
//    res = 0.;
//    updateCoeff[cellID] = 0.;
//
//    CellData::Itr cell = cells.getItr(cellID);   
//    const CFuint nbFacesInCell = cell.getNbActiveFacesInCell();
//    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
//      const CFint stype = cell.getNeighborType(f);
//      
//      if (stype != 0) { // skip all partition faces
//	const CFuint stateID =  cell.getNeighborID(f);
//	setFluxData(f, stype, stateID, cellID, &kd, currFd, cellFaces);
//	
//	// compute face quadrature points (centroid)
//	CFreal* faceCenters = &midFaceCoord[f*PHYS::DIM];
//	computeFaceCentroid<PHYS>(&cell, f, nodes, faceCenters);
//	
//	// extrapolate solution on quadrature points on both sides of the face
//	polyRec.extrapolateOnFace(currFd, faceCenters, uX, uY, uZ, limiter);
//        fluxScheme.prepareComputation(currFd, &pmodel);
//	fluxScheme(currFd, &pmodel); // compute the convective flux across the face
//	
//	for (CFuint iEq = 0; iEq < PHYS::NBEQS; ++iEq) {
//	  const CFreal value = currFd->getResidual()[iEq];
//	  res[iEq] -= value;  // update the residual 
//	}
//	
//	// update the update coefficient
//	updateCoeff[cellID] += currFd->getUpdateCoeff();
//      }
//    }
//  }
//#ifdef CF_HAVE_OMP
//} 
//#endif
//}

//////////////////////////////////////////////////////////////////////////////

template <typename SCHEME, typename PHYSICS, CFuint NB_BLOCK_THREADS>
void ConvRHSFluxReconstructionCUDA<SCHEME,PHYSICS,NB_BLOCK_THREADS>::execute()
{
  using namespace COOLFluiD::Framework;
  using namespace COOLFluiD::Common;
  
  CFTRACEBEGIN;
  
  CFLog(VERBOSE, "ConvRHSFluxReconstructionCUDA::execute() START\n");
  
  // get the elementTypeData
  SafePtr< vector<ElementTypeData> > elemType = MeshDataStack::getActive()->getElementTypeData();

  // get InnerCells TopologicalRegionSet
  SafePtr<TopologicalRegionSet> cells = MeshDataStack::getActive()->getTrs("InnerCells");

  // get the geodata of the geometric entity builder and set the TRS
  StdTrsGeoBuilder::GeoData& geoDataCell = m_cellBuilder->getDataGE();
  geoDataCell.trs = cells;
  
  // get InnerFaces TopologicalRegionSet
  SafePtr<TopologicalRegionSet> faces = MeshDataStack::getActive()->getTrs("InnerFaces");

  // get the face start indexes
  vector< CFuint >& innerFacesStartIdxs = getMethodData().getInnerFacesStartIdxs();

  // get number of face orientations
  const CFuint nbrFaceOrients = innerFacesStartIdxs.size()-1;

  // get the geodata of the face builder and set the TRSs
  FaceToCellGEBuilder::GeoData& geoDataFace = m_faceBuilder->getDataGE();
  geoDataFace.cellsTRS = cells;
  geoDataFace.facesTRS = faces;
  geoDataFace.isBoundary = false;
  
  // loop over element types, for the moment there should only be one
  const CFuint nbrElemTypes = elemType->size();
  cf_assert(nbrElemTypes == 1);
  
  // get start and end indexes for this type of element
  cf_assert((*elemType)[0].getStartIdx() == 0);
  const CFuint nbCells   = (*elemType)[0].getEndIdx();
  cf_assert(nbCells > 0);
  
  initializeComputationRHS();

  const CFuint nbStates = socket_states.getDataHandle().size();
  cf_assert(nbStates > 0);
  DataHandle<CFreal> updateCoeff = socket_updateCoeff.getDataHandle();
  DataHandle<CFreal> rhs = socket_rhs.getDataHandle(); 
  DataHandle<CFreal> solPntNormals = socket_solPntNormals.getDataHandle(); 
  
  SafePtr<SCHEME> lf  = getMethodData().getRiemannFlux().d_castTo<SCHEME>();
  SafePtr<typename PHYSICS::PTERM> phys = PhysicalModelStack::getActive()->getImplementor()->
    getConvectiveTerm().d_castTo<typename PHYSICS::PTERM>();
  
#ifdef CF_HAVE_CUDA
  typedef typename SCHEME::template DeviceFunc<GPU, PHYSICS> FluxScheme;  
#else
  typedef typename SCHEME::template DeviceFunc<CPU, PHYSICS> FluxScheme;
#endif 
  
  if (m_onGPU) {
#ifdef CF_HAVE_CUDA

    CudaEnv::CudaTimer& timer = CudaEnv::CudaTimer::getInstance();
    timer.start();
    
    // copy of data that change at every iteration
    socket_states.getDataHandle().getGlobalArray()->put(); 
    socket_rhs.getDataHandle().getLocalArray()->put(); 
CFLog(INFO, "nb normals: " << socket_solPntNormals.getDataHandle().size() << ", n0: " << socket_solPntNormals.getDataHandle()[0] << "\n");
socket_solPntNormals.getDataHandle().getLocalArray()->put();
    DataHandle<Framework::State*, Framework::GLOBAL > statesI = socket_states.getDataHandle();
    
    CFLog(INFO, "state: " << *(statesI[0]) << "\n");
     
    CFLog(VERBOSE, "ConvRHSFluxReconstructionCUDA::execute() => CPU-->GPU data transfer took " << timer.elapsed() << " s\n");
    timer.start();
    
    ConfigOptionPtr<SCHEME,  NOTYPE, GPU> dcof(lf);
    ConfigOptionPtr<typename PHYSICS::PTERM, NOTYPE, GPU> dcop(phys);

    const CFuint blocksPerGrid = CudaEnv::CudaDeviceManager::getInstance().getBlocksPerGrid(nbCells);
    const CFuint nThreads = CudaEnv::CudaDeviceManager::getInstance().getNThreads();
    CFLog(VERBOSE, "blocksPerGrid: " << blocksPerGrid << ", threads: " << nThreads << "\n");

    //dim3 blocks(m_nbBlocksPerGridX, m_nbBlocksPerGridY);
    
    //hipFuncSetCacheConfig(reinterpret_cast<const void*>("computeGradientsKernel"), hipFuncCachePreferL1);
    
    // hipFuncSetCacheConfig(reinterpret_cast<const void*>("computeFluxKernel"), hipFuncCachePreferL1);

    // compute the convective flux in each cell
    computeStateLocalRHSKernel<FluxScheme,PHYSICS> <<<blocksPerGrid,nThreads>>> 
      (dcof.getPtr(),
       dcop.getPtr(),
       nbCells,
       socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
       updateCoeff.getLocalArray()->ptrDev(), 
       rhs.getLocalArray()->ptrDev(),
       solPntNormals.getLocalArray()->ptrDev(),
       m_nbrSolPnts,
       4,
       m_cellInfo.ptrDev(),
       m_stateIDs.ptrDev(),
       m_neighbCellIDs.ptrDev(),
       m_dim,
       m_nbrEqs,
       m_nbrFlxPnts,
       m_nbrSolSolDep,
       m_solSolDep2.ptrDev(),
       m_nbrFlxDep,
       m_solFlxDep2.ptrDev(),
       m_solPolyDerivAtSolPnts2.ptrDev(),
       m_solPolyValsAtFlxPnts2.ptrDev(),
       m_flxPntFlxDim2.ptrDev(),
       m_corrFctDiv2.ptrDev());
    hipDeviceSynchronize();
    
    CFLog(VERBOSE, "ConvRHSFluxReconstructionCUDA::execute() => computeFluxKernel took " << timer.elapsed() << " s\n");
    
    timer.start();
    rhs.getLocalArray()->get();
    updateCoeff.getLocalArray()->get();
    CFLog(VERBOSE, "ConvRHSFluxReconstructionCUDA::execute() => GPU-->CPU data transfer took " << timer.elapsed() << " s\n");

#endif
}
  else {
      for (CFuint i = 0; i < nbCells; i++)
      {
          CFreal ID = i*5+i*20;
      }
    // AL: useful fo debugging
    // for (CFuint i = 0; i <  m_ghostStates.size()/9; ++i) {
    //   std::cout.precision(12); std::cout << "g" << i << " => ";
    //   for (CFuint j = 0; j < 9; ++j) {
    // 	std::cout << m_ghostStates[i*9+j] << " ";
    //   }
    //   std::cout << "\n";
    // }
    // for (CFuint i = 0; i <  socket_states.getDataHandle().size(); ++i) {
    //   std::cout.precision(12); std::cout << i << " => "<< *socket_states.getDataHandle()[i] <<"\n";
    // }
    
//    ConfigOptionPtr<SCHEME>  dcof(lf);
//    ConfigOptionPtr<POLYREC> dcor(pr);
//    ConfigOptionPtr<LIMITER> dcol(lm);
//    ConfigOptionPtr<typename PHYSICS::PTERM> dcop(phys);
//    
//    computeFluxCPU<FluxScheme, PolyRec, Limiter>
//      (m_nbThreadsOMP,
//       dcof.getPtr(),
//       dcor.getPtr(),
//       dcol.getPtr(),
//       dcop.getPtr(),
//       nbCells,
//       socket_states.getDataHandle().getGlobalArray()->ptr(), 
//       socket_nodes.getDataHandle().getGlobalArray()->ptr(),
//       m_centerNodes.ptr(), 
//       m_ghostStates.ptr(),
//       m_ghostNodes.ptr(),
//       socket_uX.getDataHandle().getLocalArray()->ptr(),
//       socket_uY.getDataHandle().getLocalArray()->ptr(),
//       socket_uZ.getDataHandle().getLocalArray()->ptr(),
//       socket_limiter.getDataHandle().getLocalArray()->ptr(),
//       updateCoeff.getLocalArray()->ptr(), 
//       rhs.getLocalArray()->ptr(),
//       normals.getLocalArray()->ptr(),
//       isOutward.getLocalArray()->ptr(),
//       m_cellInfo.ptr(),
//       m_cellStencil.ptr(),
//       m_cellFaces->getPtr()->ptr(),
//       m_cellNodes->getPtr()->ptr(),
//       m_neighborTypes.ptr(),
//       m_cellConn.ptr());
  }
  
// for (int i = 0; i < updateCoeff.size(); ++i) {
//      std::cout << "updateCoeff[" << i << "] = " << updateCoeff[i]  << std::endl;
//       /* std::cout << "rhs[" << i << "] = ";
//        for (int j = 0; j < 9; ++j) {
//          std::cout << rhs[i*9+j] << " ";
//        }
//        std::cout << std::endl;*/
// } 
//   abort();
  // for (;;) {}
  
  //finalizeComputationRHS();
  
  CFLog(VERBOSE, "ConvRHSFluxReconstructionCUDA::execute() END\n");
  
  CFTRACEEND;
}

//////////////////////////////////////////////////////////////////////////////

    } // namespace FluxReconstructionMethod

} // namespace COOLFluiD